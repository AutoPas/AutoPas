#include "hip/hip_runtime.h"
/**
 * @file LJFunctorCuda.cu
 *
 * @date 9.1.2019
 * @author jspahl
 */
#include <iostream>
#include "autopas/pairwiseFunctors/LJFunctorCuda.cuh"
#include "autopas/utils/CudaExceptionHandler.h"
#include "autopas/utils/ExceptionHandler.h"
#include "hip/hip_math_constants.h"

namespace autopas {

#define CREATESWITCHCASE(blockSize, gridSize, function, params)                \
  case blockSize:                                                              \
    function<floatType, blockSize><<<gridSize, blockSize, 0, stream>>> params; \
    break;

#define CREATESWITCHCASES(gridSize, function, params) \
  CREATESWITCHCASE(32, gridSize, function, params)    \
  CREATESWITCHCASE(64, gridSize, function, params)    \
  CREATESWITCHCASE(96, gridSize, function, params)    \
  CREATESWITCHCASE(128, gridSize, function, params)   \
  CREATESWITCHCASE(160, gridSize, function, params)   \
  CREATESWITCHCASE(192, gridSize, function, params)   \
  CREATESWITCHCASE(224, gridSize, function, params)   \
  CREATESWITCHCASE(256, gridSize, function, params)   \
  CREATESWITCHCASE(288, gridSize, function, params)   \
  CREATESWITCHCASE(320, gridSize, function, params)   \
  CREATESWITCHCASE(352, gridSize, function, params)   \
  CREATESWITCHCASE(384, gridSize, function, params)   \
  CREATESWITCHCASE(416, gridSize, function, params)   \
  CREATESWITCHCASE(448, gridSize, function, params)   \
  CREATESWITCHCASE(480, gridSize, function, params)   \
  CREATESWITCHCASE(512, gridSize, function, params)   \
  CREATESWITCHCASE(544, gridSize, function, params)   \
  CREATESWITCHCASE(576, gridSize, function, params)   \
  CREATESWITCHCASE(608, gridSize, function, params)   \
  CREATESWITCHCASE(640, gridSize, function, params)   \
  CREATESWITCHCASE(672, gridSize, function, params)   \
  CREATESWITCHCASE(704, gridSize, function, params)   \
  CREATESWITCHCASE(736, gridSize, function, params)   \
  CREATESWITCHCASE(768, gridSize, function, params)   \
  CREATESWITCHCASE(800, gridSize, function, params)   \
  CREATESWITCHCASE(832, gridSize, function, params)   \
  CREATESWITCHCASE(864, gridSize, function, params)   \
  CREATESWITCHCASE(896, gridSize, function, params)   \
  CREATESWITCHCASE(928, gridSize, function, params)   \
  CREATESWITCHCASE(960, gridSize, function, params)   \
  CREATESWITCHCASE(992, gridSize, function, params)   \
  CREATESWITCHCASE(1024, gridSize, function, params)

__constant__ LJFunctorConstants<float> global_constants_float;
__constant__ LJFunctorConstants<double> global_constants_double;

__constant__ unsigned int linkedCellsOffsetsSize;
__constant__ int linkedCellsOffsets[27];

template <typename T>
__device__ inline LJFunctorConstants<T>& getConstants() {
  return global_constants_float;
}
template <>
__device__ inline LJFunctorConstants<double>& getConstants<double>() {
  return global_constants_double;
}

template <typename T>
__device__ inline T getInfinity() {
  return HIP_INF_F;
}
template <>
__device__ inline double getInfinity<double>() {
  return HIP_INF;
}

template <typename floatType>
__device__ inline typename vec3<floatType>::Type bodyBodyF(typename vec3<floatType>::Type i,
                                                           typename vec3<floatType>::Type j,
                                                           typename vec3<floatType>::Type fi) {
  floatType drx = i.x - j.x;
  floatType drz = i.z - j.z;
  floatType dry = i.y - j.y;

  floatType dr2 = drx * drx + dry * dry + drz * drz;

  if (dr2 > getConstants<floatType>().cutoffsquare | dr2 == 0.0) {
    return fi;
  }

  floatType invdr2 = 1. / dr2;
  floatType lj6 = getConstants<floatType>().sigmasquare * invdr2;
  lj6 = lj6 * lj6 * lj6;
  floatType lj12 = lj6 * lj6;
  floatType lj12m6 = lj12 - lj6;
  floatType fac = getConstants<floatType>().epsilon24 * (lj12 + lj12m6) * invdr2;

  fi.x += drx * fac;
  fi.y += dry * fac;
  fi.z += drz * fac;

  return fi;
}

template <typename floatType, bool n3AdditionSafe = false>
__device__ inline typename vec3<floatType>::Type bodyBodyFN3(typename vec3<floatType>::Type i,
                                                             typename vec3<floatType>::Type j,
                                                             typename vec3<floatType>::Type fi,
                                                             typename vec3<floatType>::Type* fj) {
  floatType drx = i.x - j.x;
  floatType drz = i.z - j.z;
  floatType dry = i.y - j.y;

  floatType dr2 = drx * drx + dry * dry + drz * drz;

  if (dr2 > getConstants<floatType>().cutoffsquare) {
    return fi;
  }

  floatType invdr2 = 1. / dr2;
  floatType lj6 = getConstants<floatType>().sigmasquare * invdr2;
  lj6 = lj6 * lj6 * lj6;
  floatType lj12 = lj6 * lj6;
  floatType lj12m6 = lj12 - lj6;
  floatType fac = getConstants<floatType>().epsilon24 * (lj12 + lj12m6) * invdr2;

  floatType dfx = drx * fac;
  floatType dfy = dry * fac;
  floatType dfz = drz * fac;

  fi.x += dfx;
  fi.y += dfy;
  fi.z += dfz;

  if (n3AdditionSafe) {
    fj->x -= dfx;
    fj->y -= dfy;
    fj->z -= dfz;
  } else {
    atomicAdd(&(fj->x), -dfx);
    atomicAdd(&(fj->y), -dfy);
    atomicAdd(&(fj->z), -dfz);
  }
  return fi;
}

template <typename floatType, int block_size>
__global__ void SoAFunctorNoN3(LJFunctorCudaSoA<floatType> cell1) {
  __shared__ typename vec3<floatType>::Type block_pos[block_size];
  int i, tile;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  typename vec3<floatType>::Type myposition = {0, 0, 0};
  typename vec3<floatType>::Type myf = {0, 0, 0};
  if (tid < cell1._size) {
    myposition.x = cell1._posX[tid];
    myposition.y = cell1._posY[tid];
    myposition.z = cell1._posZ[tid];
  }

  for (i = block_size, tile = 0; i < cell1._size; i += block_size, ++tile) {
    int idx = tile * block_size + threadIdx.x;

    block_pos[threadIdx.x] = {cell1._posX[idx], cell1._posY[idx], cell1._posZ[idx]};
    __syncthreads();
    if (tid < cell1._size) {
      for (int j = 0; j < blockDim.x; ++j) {
        myf = bodyBodyF<floatType>(myposition, block_pos[j], myf);
      }
    }
    __syncthreads();
  }
  {
    int idx = tile * block_size + threadIdx.x;
    block_pos[threadIdx.x] = {cell1._posX[idx], cell1._posY[idx], cell1._posZ[idx]};
    __syncthreads();

    const int size = cell1._size - tile * blockDim.x;
    for (int j = 0; j < size; ++j) {
      myf = bodyBodyF<floatType>(myposition, block_pos[j], myf);
    }

    __syncthreads();
  }

  atomicAdd(cell1._forceX + tid, myf.x);
  atomicAdd(cell1._forceY + tid, myf.y);
  atomicAdd(cell1._forceZ + tid, myf.z);
}

template <typename floatType, int block_size>
__global__ void SoAFunctorNoN3Pair(LJFunctorCudaSoA<floatType> cell1, LJFunctorCudaSoA<floatType> cell2) {
  __shared__ typename vec3<floatType>::Type block_pos[block_size];
  int i, tile;
  int tid = blockIdx.x * block_size + threadIdx.x;
  typename vec3<floatType>::Type myposition;
  typename vec3<floatType>::Type myf = {0, 0, 0};

  if (tid < cell1._size) {
    myposition.x = cell1._posX[tid];
    myposition.y = cell1._posY[tid];
    myposition.z = cell1._posZ[tid];
  }

  for (i = 0, tile = 0; i < cell2._size; i += block_size, ++tile) {
    int idx = tile * block_size + threadIdx.x;

    if (idx < cell2._size) block_pos[threadIdx.x] = {cell2._posX[idx], cell2._posY[idx], cell2._posZ[idx]};
    __syncthreads();

    const int size = min(block_size, cell2._size - i);
    for (int j = 0; j < size; ++j) {
      myf = bodyBodyF<floatType>(myposition, block_pos[j], myf);
    }
    __syncthreads();
  }
  atomicAdd(cell1._forceX + tid, myf.x);
  atomicAdd(cell1._forceY + tid, myf.y);
  atomicAdd(cell1._forceZ + tid, myf.z);
}

template <typename floatType, int block_size, bool NMisMultipleBlockSize = false>
__global__ void SoAFunctorN3(LJFunctorCudaSoA<floatType> cell1) {
  __shared__ typename vec3<floatType>::Type cell1_pos_shared[block_size];
  __shared__ typename vec3<floatType>::Type cell1_forces_shared[block_size];
  int tid = blockIdx.x * block_size + threadIdx.x;
  typename vec3<floatType>::Type myposition = {getInfinity<floatType>(), getInfinity<floatType>(),
                                               getInfinity<floatType>()};
  typename vec3<floatType>::Type myf = {0, 0, 0};
  int i, tile;
  const int mask = block_size - 1;

  if (not NMisMultipleBlockSize && tid < cell1._size) {
    myposition.x = cell1._posX[tid];
    myposition.y = cell1._posY[tid];
    myposition.z = cell1._posZ[tid];
  }

  for (i = 0, tile = 0; tile < blockIdx.x; i += block_size, ++tile) {
    int idx = tile * block_size + threadIdx.x;
    cell1_pos_shared[threadIdx.x] = {cell1._posX[idx], cell1._posY[idx], cell1._posZ[idx]};
    cell1_forces_shared[threadIdx.x] = {0, 0, 0};
    __syncthreads();

    for (int j = 0; j < block_size; ++j) {
      unsigned int offset;
      if ((block_size & (block_size - 1)) == 0) {
        offset = (j + threadIdx.x) & mask;
      } else {
        offset = (j + threadIdx.x) % mask;
      }
      myf = bodyBodyFN3<floatType>(myposition, cell1_pos_shared[offset], myf, cell1_forces_shared + offset);
    }
    __syncthreads();

    atomicAdd(cell1._forceX + idx, cell1_forces_shared[threadIdx.x].x);
    atomicAdd(cell1._forceY + idx, cell1_forces_shared[threadIdx.x].y);
    atomicAdd(cell1._forceZ + idx, cell1_forces_shared[threadIdx.x].z);
    __syncthreads();
  }

  {
    int idx = blockIdx.x * block_size + threadIdx.x;
    cell1_pos_shared[threadIdx.x] = {cell1._posX[idx], cell1._posY[idx], cell1._posZ[idx]};
    cell1_forces_shared[threadIdx.x] = {0, 0, 0};
    __syncthreads();

    for (int j = threadIdx.x - 1; j >= 0; --j) {
      myf = bodyBodyFN3<floatType>(myposition, cell1_pos_shared[j], myf, cell1_forces_shared + j);
    }
    __syncthreads();

    atomicAdd(cell1._forceX + idx, cell1_forces_shared[threadIdx.x].x);
    atomicAdd(cell1._forceY + idx, cell1_forces_shared[threadIdx.x].y);
    atomicAdd(cell1._forceZ + idx, cell1_forces_shared[threadIdx.x].z);
    __syncthreads();
  }

  atomicAdd(cell1._forceX + tid, myf.x);
  atomicAdd(cell1._forceY + tid, myf.y);
  atomicAdd(cell1._forceZ + tid, myf.z);
}

template <typename floatType, int block_size, bool NMisMultipleBlockSize = false>
__global__ void SoAFunctorN3Pair(LJFunctorCudaSoA<floatType> cell1, LJFunctorCudaSoA<floatType> cell2) {
  __shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
  __shared__ typename vec3<floatType>::Type cell2_forces_shared[block_size];
  int tid = blockIdx.x * block_size + threadIdx.x;
  typename vec3<floatType>::Type myposition = {getInfinity<floatType>(), getInfinity<floatType>(),
                                               getInfinity<floatType>()};
  typename vec3<floatType>::Type myf = {0, 0, 0};
  int i, tile;
  const int mask = block_size - 1;

  if (not NMisMultipleBlockSize && tid < cell1._size) {
    myposition.x = cell1._posX[tid];
    myposition.y = cell1._posY[tid];
    myposition.z = cell1._posZ[tid];
  }
  for (i = block_size, tile = 0; i <= cell2._size; i += block_size, ++tile) {
    int idx = tile * block_size + threadIdx.x;
    cell2_pos_shared[threadIdx.x] = {cell2._posX[idx], cell2._posY[idx], cell2._posZ[idx]};
    cell2_forces_shared[threadIdx.x] = {0, 0, 0};
    __syncthreads();

    for (int j = 0; j < block_size; ++j) {
      unsigned int offset;
      if ((block_size & (block_size - 1)) == 0) {
        offset = (j + threadIdx.x) & mask;
      } else {
        offset = (j + threadIdx.x) % mask;
      }
      myf = bodyBodyFN3<floatType, true>(myposition, cell2_pos_shared[offset], myf, cell2_forces_shared + offset);
    }
    __syncthreads();

    atomicAdd(cell2._forceX + idx, cell2_forces_shared[threadIdx.x].x);
    atomicAdd(cell2._forceY + idx, cell2_forces_shared[threadIdx.x].y);
    atomicAdd(cell2._forceZ + idx, cell2_forces_shared[threadIdx.x].z);
    __syncthreads();
  }
  if ((not NMisMultipleBlockSize) && (i > cell2._size)) {
    int idx = tile * block_size + threadIdx.x;
    if (idx < cell2._size) {
      cell2_pos_shared[threadIdx.x] = {cell2._posX[idx], cell2._posY[idx], cell2._posZ[idx]};
      cell2_forces_shared[threadIdx.x] = {0, 0, 0};
    }
    __syncthreads();

    const int size = block_size + cell2._size - i;
    for (int j = 0; j < size; ++j) {
      const int offset = (j + threadIdx.x) % size;
      myf = bodyBodyFN3<floatType>(myposition, cell2_pos_shared[offset], myf, cell2_forces_shared + offset);
    }
    __syncthreads();
    if (idx < cell2._size) {
      atomicAdd(cell2._forceX + idx, cell2_forces_shared[threadIdx.x].x);
      atomicAdd(cell2._forceY + idx, cell2_forces_shared[threadIdx.x].y);
      atomicAdd(cell2._forceZ + idx, cell2_forces_shared[threadIdx.x].z);
      __syncthreads();
    }
  }

  atomicAdd(cell1._forceX + tid, myf.x);
  atomicAdd(cell1._forceY + tid, myf.y);
  atomicAdd(cell1._forceZ + tid, myf.z);
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::SoAFunctorNoN3Wrapper(FunctorCudaSoA<floatType>* cell1Base, hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType>*>(cell1Base);

  switch (_num_threads) {
    CREATESWITCHCASES(numRequiredBlocks(cell1._size), SoAFunctorNoN3, (cell1));
    default:
      autopas::utils::ExceptionHandler::exception(std::string("cuda Kernel size not available"));
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::SoAFunctorNoN3PairWrapper(FunctorCudaSoA<floatType>* cell1Base,
                                                                FunctorCudaSoA<floatType>* cell2Base,
                                                                hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType>*>(cell1Base);
  LJFunctorCudaSoA<floatType> cell2 = *static_cast<LJFunctorCudaSoA<floatType>*>(cell2Base);

  switch (_num_threads) {
    CREATESWITCHCASES(numRequiredBlocks(cell1._size), SoAFunctorNoN3Pair, (cell1, cell2));
    default:
      autopas::utils::ExceptionHandler::exception(std::string("cuda Kernel size not available"));
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::SoAFunctorN3Wrapper(FunctorCudaSoA<floatType>* cell1Base, hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType>*>(cell1Base);

  switch (_num_threads) {
    CREATESWITCHCASES(numRequiredBlocks(cell1._size), SoAFunctorN3, (cell1));
    default:
      autopas::utils::ExceptionHandler::exception(std::string("cuda Kernel size not available"));
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::SoAFunctorN3PairWrapper(FunctorCudaSoA<floatType>* cell1Base,
                                                              FunctorCudaSoA<floatType>* cell2Base,
                                                              hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType>*>(cell1Base);
  LJFunctorCudaSoA<floatType> cell2 = *static_cast<LJFunctorCudaSoA<floatType>*>(cell2Base);

  switch (_num_threads) {
    CREATESWITCHCASES(numRequiredBlocks(cell1._size), SoAFunctorN3Pair, (cell1, cell2));
    default:
      autopas::utils::ExceptionHandler::exception(std::string("cuda Kernel size not available"));
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <typename floatType, int block_size>
__global__ void LinkedCellsTraversalNoN3(LJFunctorCudaSoA<floatType> cell, unsigned int* cids, size_t* cellSizes) {
  int own_cid = cids[blockIdx.x];
  __shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
  typename vec3<floatType>::Type myposition = {getInfinity<floatType>(), getInfinity<floatType>(),
                                               getInfinity<floatType>()};
  typename vec3<floatType>::Type myf = {0, 0, 0};

  int index = cellSizes[own_cid] + threadIdx.x;
  if (threadIdx.x < (cellSizes[own_cid + 1] - cellSizes[own_cid])) {
    myposition.x = cell._posX[index];
    myposition.y = cell._posY[index];
    myposition.z = cell._posZ[index];
  }

  // other cells
  for (auto other_index = 0; other_index < linkedCellsOffsetsSize; ++other_index) {
    const int other_id = own_cid + linkedCellsOffsets[other_index];
    const size_t cell2Start = cellSizes[other_id];
    const unsigned int sizeCell2 = cellSizes[other_id + 1] - cell2Start;

    cell2_pos_shared[threadIdx.x] = {cell._posX[cell2Start + threadIdx.x], cell._posY[cell2Start + threadIdx.x],
                                     cell._posZ[cell2Start + threadIdx.x]};
    __syncthreads();
    for (int j = 0; j < sizeCell2; ++j) {
      myf = bodyBodyF<floatType>(myposition, cell2_pos_shared[j], myf);
    }
    __syncthreads();
  }
  if (threadIdx.x < (cellSizes[own_cid + 1] - cellSizes[own_cid])) {
    atomicAdd(cell._forceX + index, myf.x);
    atomicAdd(cell._forceY + index, myf.y);
    atomicAdd(cell._forceZ + index, myf.z);
  }
}

template <typename floatType, int block_size>
__global__ void LinkedCellsTraversalN3(LJFunctorCudaSoA<floatType> cell, unsigned int* cids, size_t* cellSizes) {
  int own_cid = cids[blockIdx.x];
  __shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
  __shared__ typename vec3<floatType>::Type cell2_forces_shared[block_size];

  typename vec3<floatType>::Type myposition = {getInfinity<floatType>(), getInfinity<floatType>(),
                                               getInfinity<floatType>()};
  typename vec3<floatType>::Type myf = {0, 0, 0};

  int index = cellSizes[own_cid] + threadIdx.x;
  if (threadIdx.x < (cellSizes[own_cid + 1] - cellSizes[own_cid])) {
    myposition.x = cell._posX[index];
    myposition.y = cell._posY[index];
    myposition.z = cell._posZ[index];
  }
  // other cells
  for (auto other_index = 0; other_index < linkedCellsOffsetsSize; ++other_index) {
    const int other_id = own_cid + linkedCellsOffsets[other_index];
    const size_t cell2Start = cellSizes[other_id];
    const int sizeCell2 = cellSizes[other_id + 1] - cell2Start;

    cell2_pos_shared[threadIdx.x] = {cell._posX[cell2Start + threadIdx.x], cell._posY[cell2Start + threadIdx.x],
                                     cell._posZ[cell2Start + threadIdx.x]};
    cell2_forces_shared[threadIdx.x] = {0, 0, 0};
    __syncthreads();
    for (int j = 0; j < sizeCell2; ++j) {
      const int offset = (j + threadIdx.x) % sizeCell2;
      myf = bodyBodyFN3<floatType, false>(myposition, cell2_pos_shared[offset], myf, cell2_forces_shared + offset);
    }
    __syncthreads();

    atomicAdd(cell._forceX + cell2Start + threadIdx.x, cell2_forces_shared[threadIdx.x].x);
    atomicAdd(cell._forceY + cell2Start + threadIdx.x, cell2_forces_shared[threadIdx.x].y);
    atomicAdd(cell._forceZ + cell2Start + threadIdx.x, cell2_forces_shared[threadIdx.x].z);
    __syncthreads();
  }
  // same cells
  {
    const int cell1Start = cellSizes[own_cid];
    const int sizeCell1 = cellSizes[own_cid + 1] - cell1Start;

    cell2_pos_shared[threadIdx.x] = {cell._posX[cell1Start + threadIdx.x], cell._posY[cell1Start + threadIdx.x],
                                     cell._posZ[cell1Start + threadIdx.x]};
    __syncthreads();
    for (int j = 0; j < sizeCell1; ++j) {
      myf = bodyBodyF<floatType>(myposition, cell2_pos_shared[j], myf);
    }
    __syncthreads();
  }
  if (threadIdx.x < (cellSizes[own_cid + 1] - cellSizes[own_cid])) {
    atomicAdd(cell._forceX + index, myf.x);
    atomicAdd(cell._forceY + index, myf.y);
    atomicAdd(cell._forceZ + index, myf.z);
  }
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::LinkedCellsTraversalNoN3Wrapper(FunctorCudaSoA<floatType>* cell1Base,
                                                                      unsigned int reqThreads, unsigned int cids_size,
                                                                      unsigned int* cids, unsigned int cellSizes_size,
                                                                      size_t* cellSizes, hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType>*>(cell1Base);

  switch (reqThreads) {
    CREATESWITCHCASES(cids_size, LinkedCellsTraversalNoN3, (cell1, cids, cellSizes));
    default:
      autopas::utils::ExceptionHandler::exception(
          "Linked Cells NoN3: cuda Kernel size not available for Linked cells available. Too many particles "
          "in a cell. Requested: {}",
          reqThreads);
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::LinkedCellsTraversalN3Wrapper(FunctorCudaSoA<floatType>* cell1Base,
                                                                    unsigned int reqThreads, unsigned int cids_size,
                                                                    unsigned int* cids, unsigned int cellSizes_size,
                                                                    size_t* cellSizes, hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType>*>(cell1Base);

  switch (reqThreads) {
    CREATESWITCHCASES(cids_size, LinkedCellsTraversalN3, (cell1, cids, cellSizes));
    default:
      autopas::utils::ExceptionHandler::exception(
          "Linked Cells N3:cuda Kernel size not available for Linked cells available. Too many particles in "
          "a cell. Requested: {}",
          reqThreads);
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <typename floatType, int block_size>
__global__ void CellVerletTraversalNoN3(LJFunctorCudaSoA<floatType> cell, const unsigned int others_size,
                                        unsigned int* other_ids) {
  __shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
  typename vec3<floatType>::Type myposition = {getInfinity<floatType>(), getInfinity<floatType>(),
                                               getInfinity<floatType>()};
  typename vec3<floatType>::Type myf = {0, 0, 0};

  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  myposition.x = cell._posX[index];
  myposition.y = cell._posY[index];
  myposition.z = cell._posZ[index];

  // other cells
  for (auto other_index = others_size * blockIdx.x; other_ids[other_index] < UINT_MAX; ++other_index) {
    const size_t cell2Start = blockDim.x * other_ids[other_index];

    const size_t own_particle = cell2Start + threadIdx.x;
    cell2_pos_shared[threadIdx.x] = {cell._posX[own_particle], cell._posY[own_particle], cell._posZ[own_particle]};
    __syncthreads();
    for (int j = 0; j < block_size; ++j) {
      myf = bodyBodyF<floatType>(myposition, cell2_pos_shared[j], myf);
    }
    __syncthreads();
  }
  atomicAdd(cell._forceX + index, myf.x);
  atomicAdd(cell._forceY + index, myf.y);
  atomicAdd(cell._forceZ + index, myf.z);
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::CellVerletTraversalNoN3Wrapper(FunctorCudaSoA<floatType>* cell1Base,
                                                                     unsigned int ncells, unsigned int clusterSize,
                                                                     unsigned int others_size, unsigned int* other_ids,
                                                                     hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType>*>(cell1Base);
  switch (clusterSize) {
    CREATESWITCHCASES(ncells, CellVerletTraversalNoN3, (cell1, others_size, other_ids));
    default:
      autopas::utils::ExceptionHandler::exception(
          "cuda Kernel size not available for Verlet cells available. Too many particles in a cell. "
          "Requested: {}",
          clusterSize);
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <typename floatType, int block_size>
__global__ void CellVerletTraversalN3(LJFunctorCudaSoA<floatType> cell, unsigned int others_size,
                                      unsigned int* other_ids) {
  const unsigned int mask = block_size - 1;

  __shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
  __shared__ typename vec3<floatType>::Type cell2_forces_shared[block_size];

  typename vec3<floatType>::Type myposition = {getInfinity<floatType>(), getInfinity<floatType>(),
                                               getInfinity<floatType>()};
  typename vec3<floatType>::Type myf = {0, 0, 0};

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  myposition.x = cell._posX[index];
  myposition.y = cell._posY[index];
  myposition.z = cell._posZ[index];

  // other cells
  for (auto other_index = others_size * blockIdx.x; other_ids[other_index] != UINT_MAX; ++other_index) {
    const unsigned int cell2Start = blockDim.x * other_ids[other_index];

    cell2_pos_shared[threadIdx.x] = {cell._posX[cell2Start + threadIdx.x], cell._posY[cell2Start + threadIdx.x],
                                     cell._posZ[cell2Start + threadIdx.x]};
    cell2_forces_shared[threadIdx.x] = {0, 0, 0};
    __syncthreads();
    for (int j = 0; j < block_size; ++j) {
      unsigned int offset = 0;
      if ((block_size & (block_size - 1)) == 0) {
        offset = (j + threadIdx.x) & mask;
      } else {
        offset = (j + threadIdx.x) % block_size;
      }
      myf = bodyBodyFN3<floatType, false>(myposition, cell2_pos_shared[offset], myf, cell2_forces_shared + offset);
    }
    __syncthreads();

    atomicAdd(cell._forceX + cell2Start + threadIdx.x, cell2_forces_shared[threadIdx.x].x);
    atomicAdd(cell._forceY + cell2Start + threadIdx.x, cell2_forces_shared[threadIdx.x].y);
    atomicAdd(cell._forceZ + cell2Start + threadIdx.x, cell2_forces_shared[threadIdx.x].z);
    __syncthreads();
  }

  // same cluster without N3
  {
    const unsigned int cellStart = blockIdx.x * blockDim.x;

    cell2_pos_shared[threadIdx.x] = {cell._posX[cellStart + threadIdx.x], cell._posY[cellStart + threadIdx.x],
                                     cell._posZ[cellStart + threadIdx.x]};
    __syncthreads();
    for (int j = 0; j < block_size; ++j) {
      myf = bodyBodyF<floatType>(myposition, cell2_pos_shared[j], myf);
    }
    __syncthreads();
  }
  atomicAdd(cell._forceX + index, myf.x);
  atomicAdd(cell._forceY + index, myf.y);
  atomicAdd(cell._forceZ + index, myf.z);
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::CellVerletTraversalN3Wrapper(FunctorCudaSoA<floatType>* cell1Base,
                                                                   unsigned int ncells, unsigned int clusterSize,
                                                                   unsigned int others_size, unsigned int* other_ids,
                                                                   hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType>*>(cell1Base);
  switch (clusterSize) {
    CREATESWITCHCASES(ncells, CellVerletTraversalN3, (cell1, others_size, other_ids));
    default:
      autopas::utils::ExceptionHandler::exception(
          "cuda Kernel size not available for Verlet cells available. Too many particles in a cell. "
          "Requested: {}",
          clusterSize);
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <>
void LJFunctorCudaWrapper<float>::loadConstants(FunctorCudaConstants<float>* constants) {
  LJFunctorConstants<float>* c = static_cast<LJFunctorConstants<float>*>(constants);

  hipMemcpyToSymbol(HIP_SYMBOL(global_constants_float), c, sizeof(LJFunctorConstants<float>));
}

template <>
void LJFunctorCudaWrapper<double>::loadConstants(FunctorCudaConstants<double>* constants) {
  LJFunctorConstants<double>* c = static_cast<LJFunctorConstants<double>*>(constants);
  autopas::utils::CudaExceptionHandler::checkErrorCode(
      hipMemcpyToSymbol(HIP_SYMBOL(global_constants_double), c, sizeof(LJFunctorConstants<double>)));
}

template <typename T>
void LJFunctorCudaWrapper<T>::loadConstants(FunctorCudaConstants<T>* constants) {
  autopas::utils::ExceptionHandler::exception("Cuda constants with unknown Type loaded");
}

template <typename T>
void LJFunctorCudaWrapper<T>::loadLinkedCellsOffsets(unsigned int offsets_size, int* offsets) {
  if (offsets_size > 27) {
    autopas::utils::ExceptionHandler::exception(
        "LJFunctorCudaWrapper does not support linked cells with >27 neighbors");
  }
  autopas::utils::CudaExceptionHandler::checkErrorCode(
      hipMemcpyToSymbol(HIP_SYMBOL(linkedCellsOffsetsSize), &offsets_size, sizeof(unsigned int)));
  autopas::utils::CudaExceptionHandler::checkErrorCode(
      hipMemcpyToSymbol(HIP_SYMBOL(linkedCellsOffsets), offsets, offsets_size * sizeof(int)));
}

template class LJFunctorCudaWrapper<float>;
template class LJFunctorCudaWrapper<double>;

}  // namespace autopas
