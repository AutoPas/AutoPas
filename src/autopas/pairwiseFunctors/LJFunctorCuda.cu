#include "hip/hip_runtime.h"
/**
 * @file LJFunctorCuda.h
 *
 * @date 9.1.2019
 * @author jspahl
 */
#include "autopas/pairwiseFunctors/LJFunctorCuda.cuh"
#include <iostream>
#include "autopas/utils/ExceptionHandler.h"
#include "autopas/utils/CudaExceptionHandler.h"
#include "hip/hip_math_constants.h"

namespace autopas {

__constant__ constants<float> global_constants_float;
__constant__ constants<double> global_constants_double;

template<typename T>
__device__ inline constants<T>& getConstants() {
	return global_constants_float;
}
template<>
__device__ inline constants<double>& getConstants<double>() {
	return global_constants_double;
}

template<typename T>
__device__ inline T getInfinity() {
	return HIP_INF_F;
}
template<>
__device__ inline double getInfinity<double>() {
	return HIP_INF;
}

template<typename floatType>
__device__
inline typename vec3<floatType>::Type bodyBodyF(
		typename vec3<floatType>::Type i, typename vec3<floatType>::Type j,
		typename vec3<floatType>::Type fi) {
	floatType drx = i.x - j.x;
	floatType dry = i.y - j.y;
	floatType drz = i.z - j.z;

	floatType dr2 = drx * drx + dry * dry + drz * drz;

	if (dr2 > getConstants<floatType>().cutoffsquare | dr2 == 0.0) {
		return fi;
	}

	floatType invdr2 = 1. / dr2;
	floatType lj6 = getConstants<floatType>().sigmasquare * invdr2;
	lj6 = lj6 * lj6 * lj6;
	floatType lj12 = lj6 * lj6;
	floatType lj12m6 = lj12 - lj6;
	floatType fac = getConstants<floatType>().epsilon24 * (lj12 + lj12m6)
			* invdr2;

	fi.x += drx * fac;
	fi.y += dry * fac;
	fi.z += drz * fac;

	return fi;
}

template<typename floatType, bool n3AdditionSafe = false>
__device__
inline typename vec3<floatType>::Type bodyBodyFN3(
		typename vec3<floatType>::Type i, typename vec3<floatType>::Type j,
		typename vec3<floatType>::Type fi, typename vec3<floatType>::Type* fj) {
	floatType drx = i.x - j.x;
	floatType dry = i.y - j.y;
	floatType drz = i.z - j.z;

	floatType dr2 = drx * drx + dry * dry + drz * drz;

	if (dr2 > getConstants<floatType>().cutoffsquare) {
		return fi;
	}

	floatType invdr2 = 1. / dr2;
	floatType lj6 = getConstants<floatType>().sigmasquare * invdr2;
	lj6 = lj6 * lj6 * lj6;
	floatType lj12 = lj6 * lj6;
	floatType lj12m6 = lj12 - lj6;
	floatType fac = getConstants<floatType>().epsilon24 * (lj12 + lj12m6)
			* invdr2;

	floatType dfx = drx * fac;
	floatType dfy = dry * fac;
	floatType dfz = drz * fac;

	fi.x += dfx;
	fi.y += dfy;
	fi.z += dfz;

	if (n3AdditionSafe) {
		fj->x -= dfx;
		fj->y -= dfy;
		fj->z -= dfz;
	} else {
		atomicAdd(&(fj->x), -dfx);
		atomicAdd(&(fj->y), -dfy);
		atomicAdd(&(fj->z), -dfz);
	}
	return fi;
}

template<typename floatType, int block_size>
__global__
void SoAFunctorNoN3(LJFunctorCudaSoA<floatType> cell1) {
	__shared__ typename vec3<floatType>::Type block_pos[block_size];
	int i, tile;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	typename vec3<floatType>::Type myposition = { 0, 0, 0 };
	typename vec3<floatType>::Type myf = { 0, 0, 0 };
	if (tid < cell1._size) {
		myposition.x = cell1._posX[tid];
		myposition.y = cell1._posY[tid];
		myposition.z = cell1._posZ[tid];
	}

	for (i = block_size, tile = 0; i < cell1._size; i += block_size, ++tile) {
		int idx = tile * block_size + threadIdx.x;

		block_pos[threadIdx.x] = {cell1._posX[idx], cell1._posY[idx], cell1._posZ[idx]};
		__syncthreads();
		if (tid < cell1._size) {
			for (int j = 0; j < blockDim.x; ++j) {
				myf = bodyBodyF<floatType>(myposition, block_pos[j], myf);
			}
		}
		__syncthreads();
	}
	{
		int idx = tile * block_size + threadIdx.x;
		block_pos[threadIdx.x] = {cell1._posX[idx], cell1._posY[idx], cell1._posZ[idx]};
		__syncthreads();

		const int size = cell1._size - tile * blockDim.x;
		for (int j = 0; j < size; ++j) {
			myf = bodyBodyF<floatType>(myposition, block_pos[j], myf);
		}

		__syncthreads();
	}

	atomicAdd(cell1._forceX + tid, myf.x);
	atomicAdd(cell1._forceY + tid, myf.y);
	atomicAdd(cell1._forceZ + tid, myf.z);
}

template<typename floatType, int block_size>
__global__
void SoAFunctorNoN3Pair(LJFunctorCudaSoA<floatType> cell1,
		LJFunctorCudaSoA<floatType> cell2) {
	__shared__ typename vec3<floatType>::Type block_pos[block_size];
	int i, tile;
	int tid = blockIdx.x * block_size + threadIdx.x;
	typename vec3<floatType>::Type myposition;
	typename vec3<floatType>::Type myf = { 0, 0, 0 };

	if (tid < cell1._size) {
		myposition.x = cell1._posX[tid];
		myposition.y = cell1._posY[tid];
		myposition.z = cell1._posZ[tid];
	}

	for (i = 0, tile = 0; i < cell2._size; i += block_size, ++tile) {
		int idx = tile * block_size + threadIdx.x;

		if (idx < cell2._size)
			block_pos[threadIdx.x] = {cell2._posX[idx], cell2._posY[idx], cell2._posZ[idx]};
		__syncthreads();

		const int size = min(block_size, cell2._size - i);
		for (int j = 0; j < size; ++j) {
			myf = bodyBodyF<floatType>(myposition, block_pos[j], myf);
		}
		__syncthreads();
	}
	atomicAdd(cell1._forceX + tid, myf.x);
	atomicAdd(cell1._forceY + tid, myf.y);
	atomicAdd(cell1._forceZ + tid, myf.z);
}

template<typename floatType, int block_size, bool NMisMultipleBlockSize = false>
__global__
void SoAFunctorN3(LJFunctorCudaSoA<floatType> cell1) {
	static_assert((block_size & (block_size - 1)) == 0, "block size must be power of 2");
	__shared__ typename vec3<floatType>::Type cell1_pos_shared[block_size];
	__shared__ typename vec3<floatType>::Type cell1_forces_shared[block_size];
	int tid = blockIdx.x * block_size + threadIdx.x;
	typename vec3<floatType>::Type myposition = { getInfinity<floatType>(),
			getInfinity<floatType>(), getInfinity<floatType>() };
	typename vec3<floatType>::Type myf = { 0, 0, 0 };
	int i, tile;
	const int mask = block_size - 1;

	if (not NMisMultipleBlockSize && tid < cell1._size) {
		myposition.x = cell1._posX[tid];
		myposition.y = cell1._posY[tid];
		myposition.z = cell1._posZ[tid];
	}

	for (i = 0, tile = 0; tile < blockIdx.x; i += block_size, ++tile) {
		int idx = tile * block_size + threadIdx.x;
		cell1_pos_shared[threadIdx.x] = {cell1._posX[idx], cell1._posY[idx], cell1._posZ[idx]};
		cell1_forces_shared[threadIdx.x] = {0,0,0};
		__syncthreads();

		for (int j = 0; j < block_size; ++j) {
			const int offset = (j + threadIdx.x) & mask;
			myf = bodyBodyFN3<floatType>(myposition, cell1_pos_shared[offset],
					myf, cell1_forces_shared + offset);
		}
		__syncthreads();

		atomicAdd(cell1._forceX + idx, cell1_forces_shared[threadIdx.x].x);
		atomicAdd(cell1._forceY + idx, cell1_forces_shared[threadIdx.x].y);
		atomicAdd(cell1._forceZ + idx, cell1_forces_shared[threadIdx.x].z);
		__syncthreads();
	}

	{
		int idx = blockIdx.x * block_size + threadIdx.x;
		cell1_pos_shared[threadIdx.x] = {cell1._posX[idx], cell1._posY[idx], cell1._posZ[idx]};
		cell1_forces_shared[threadIdx.x] = {0,0,0};
		__syncthreads();

		for (int j = threadIdx.x - 1; j >= 0; --j) {
			myf = bodyBodyFN3<floatType>(myposition, cell1_pos_shared[j], myf,
					cell1_forces_shared + j);
		}
		__syncthreads();

		atomicAdd(cell1._forceX + idx, cell1_forces_shared[threadIdx.x].x);
		atomicAdd(cell1._forceY + idx, cell1_forces_shared[threadIdx.x].y);
		atomicAdd(cell1._forceZ + idx, cell1_forces_shared[threadIdx.x].z);
		__syncthreads();
	}

	atomicAdd(cell1._forceX + tid, myf.x);
	atomicAdd(cell1._forceY + tid, myf.y);
	atomicAdd(cell1._forceZ + tid, myf.z);
}

template<typename floatType, int block_size, bool NMisMultipleBlockSize = false>
__global__
void SoAFunctorN3Pair(LJFunctorCudaSoA<floatType> cell1,
		LJFunctorCudaSoA<floatType> cell2) {
	static_assert((block_size & (block_size - 1)) == 0, "block size must be power of 2");
	__shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
	__shared__ typename vec3<floatType>::Type cell2_forces_shared[block_size];
	int tid = blockIdx.x * block_size + threadIdx.x;
	typename vec3<floatType>::Type myposition = { getInfinity<floatType>(),
			getInfinity<floatType>(), getInfinity<floatType>() };
	typename vec3<floatType>::Type myf = { 0, 0, 0 };
	int i, tile;
	const int mask = block_size - 1;

	if (not NMisMultipleBlockSize && tid < cell1._size) {
		myposition.x = cell1._posX[tid];
		myposition.y = cell1._posY[tid];
		myposition.z = cell1._posZ[tid];
	}
	for (i = block_size, tile = 0; i <= cell2._size; i += block_size, ++tile) {
		int idx = tile * block_size + threadIdx.x;
		cell2_pos_shared[threadIdx.x] = {cell2._posX[idx], cell2._posY[idx], cell2._posZ[idx]};
		cell2_forces_shared[threadIdx.x] = {0,0,0};
		__syncthreads();

		for (int j = 0; j < block_size; ++j) {
			const int offset = (j + threadIdx.x) & mask;
			myf = bodyBodyFN3<floatType, true>(myposition,
					cell2_pos_shared[offset], myf,
					cell2_forces_shared + offset);
		}
		__syncthreads();

		atomicAdd(cell2._forceX + idx, cell2_forces_shared[threadIdx.x].x);
		atomicAdd(cell2._forceY + idx, cell2_forces_shared[threadIdx.x].y);
		atomicAdd(cell2._forceZ + idx, cell2_forces_shared[threadIdx.x].z);
		__syncthreads();
	}
	if ((not NMisMultipleBlockSize) && (i > cell2._size)) {
		int idx = tile * block_size + threadIdx.x;
		if (idx < cell2._size) {
			cell2_pos_shared[threadIdx.x] = {cell2._posX[idx], cell2._posY[idx], cell2._posZ[idx]};
			cell2_forces_shared[threadIdx.x] = {0,0,0};
		}
		__syncthreads();

		const int size = block_size + cell2._size - i;
		for (int j = 0; j < size; ++j) {
			const int offset = (j + threadIdx.x) % size;
			myf = bodyBodyFN3<floatType>(myposition, cell2_pos_shared[offset],
					myf, cell2_forces_shared + offset);
		}
		__syncthreads();
		if (idx < cell2._size) {
			atomicAdd(cell2._forceX + idx, cell2_forces_shared[threadIdx.x].x);
			atomicAdd(cell2._forceY + idx, cell2_forces_shared[threadIdx.x].y);
			atomicAdd(cell2._forceZ + idx, cell2_forces_shared[threadIdx.x].z);
			__syncthreads();
		}
	}

	atomicAdd(cell1._forceX + tid, myf.x);
	atomicAdd(cell1._forceY + tid, myf.y);
	atomicAdd(cell1._forceZ + tid, myf.z);
}

template<typename floatType>
void CudaWrapper::SoAFunctorNoN3Wrapper(LJFunctorCudaSoA<floatType> cell1,
		hipStream_t stream) {
	switch (_num_threads) {
	case 32:
		SoAFunctorNoN3<floatType, 32> <<<numRequiredBlocks(cell1._size), 32, 0,
				stream>>>(cell1);
		break;
	case 64:
		SoAFunctorNoN3<floatType, 64> <<<numRequiredBlocks(cell1._size), 64, 0,
				stream>>>(cell1);
		break;
	case 96:
		SoAFunctorNoN3<floatType, 96> <<<numRequiredBlocks(cell1._size), 96, 0,
				stream>>>(cell1);
		break;
	case 128:
		SoAFunctorNoN3<floatType, 128> <<<numRequiredBlocks(cell1._size), 128,
				0, stream>>>(cell1);
		break;
	case 256:
		SoAFunctorNoN3<floatType, 256> <<<numRequiredBlocks(cell1._size), 256,
				0, stream>>>(cell1);
		break;
	case 512:
		SoAFunctorNoN3<floatType, 512> <<<numRequiredBlocks(cell1._size), 512,
				0, stream>>>(cell1);
		break;
	case 1024:
		SoAFunctorNoN3<floatType, 1024> <<<numRequiredBlocks(cell1._size), 1024,
				0, stream>>>(cell1);
		break;
	default:
		autopas::utils::ExceptionHandler::exception(
				std::string("cuda Kernel size not available"));
		break;
	}
	autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template void CudaWrapper::SoAFunctorNoN3Wrapper<float>(
		LJFunctorCudaSoA<float> cell1, hipStream_t stream);
template void CudaWrapper::SoAFunctorNoN3Wrapper<double>(
		LJFunctorCudaSoA<double> cell1, hipStream_t stream);

template<typename floatType>
void CudaWrapper::SoAFunctorNoN3PairWrapper(LJFunctorCudaSoA<floatType> cell1,
		LJFunctorCudaSoA<floatType> cell2, hipStream_t stream) {
	switch (_num_threads) {
	case 32:
		SoAFunctorNoN3Pair<floatType, 32> <<<numRequiredBlocks(cell1._size), 32,
				0, stream>>>(cell1, cell2);
		break;
	case 64:
		SoAFunctorNoN3Pair<floatType, 64> <<<numRequiredBlocks(cell1._size), 64,
				0, stream>>>(cell1, cell2);
		break;
	case 96:
		SoAFunctorNoN3Pair<floatType, 96> <<<numRequiredBlocks(cell1._size), 96,
				0, stream>>>(cell1, cell2);
		break;
	case 128:
		SoAFunctorNoN3Pair<floatType, 128> <<<numRequiredBlocks(cell1._size),
				128, 0, stream>>>(cell1, cell2);
		break;
	case 256:
		SoAFunctorNoN3Pair<floatType, 256> <<<numRequiredBlocks(cell1._size),
				256, 0, stream>>>(cell1, cell2);
		break;
	case 512:
		SoAFunctorNoN3Pair<floatType, 512> <<<numRequiredBlocks(cell1._size),
				512, 0, stream>>>(cell1, cell2);
		break;
	case 1024:
		SoAFunctorNoN3Pair<floatType, 1024> <<<numRequiredBlocks(cell1._size),
				1024, 0, stream>>>(cell1, cell2);
		break;
	default:
		autopas::utils::ExceptionHandler::exception(
				std::string("cuda Kernel size not available"));
		break;
	}
	autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template void CudaWrapper::SoAFunctorNoN3PairWrapper<float>(
		LJFunctorCudaSoA<float> cell1, LJFunctorCudaSoA<float> cell2,
		hipStream_t stream);
template void CudaWrapper::SoAFunctorNoN3PairWrapper<double>(
		LJFunctorCudaSoA<double> cell1, LJFunctorCudaSoA<double> cell2,
		hipStream_t stream);

template<typename floatType>
void CudaWrapper::SoAFunctorN3Wrapper(LJFunctorCudaSoA<floatType> cell1,
		hipStream_t stream) {
	switch (_num_threads) {
	case 32:
		SoAFunctorN3<floatType, 32> <<<numRequiredBlocks(cell1._size), 32, 0,
				stream>>>(cell1);
		break;
	case 64:
		SoAFunctorN3<floatType, 64> <<<numRequiredBlocks(cell1._size), 64, 0,
				stream>>>(cell1);
		break;
	case 128:
		SoAFunctorN3<floatType, 128> <<<numRequiredBlocks(cell1._size), 128, 0,
				stream>>>(cell1);
		break;
	case 256:
		SoAFunctorN3<floatType, 256> <<<numRequiredBlocks(cell1._size), 256, 0,
				stream>>>(cell1);
		break;
	case 512:
		SoAFunctorN3<floatType, 512> <<<numRequiredBlocks(cell1._size), 512, 0,
				stream>>>(cell1);
		break;
	case 1024:
		SoAFunctorN3<floatType, 1024> <<<numRequiredBlocks(cell1._size), 1024,
				0, stream>>>(cell1);
		break;
	default:
		autopas::utils::ExceptionHandler::exception(
				std::string("cuda Kernel size not available"));
		break;
	}
	autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template void CudaWrapper::SoAFunctorN3Wrapper<float>(
		LJFunctorCudaSoA<float> cell1, hipStream_t stream);
template void CudaWrapper::SoAFunctorN3Wrapper<double>(
		LJFunctorCudaSoA<double> cell1, hipStream_t stream);

template<typename floatType>
void CudaWrapper::SoAFunctorN3PairWrapper(LJFunctorCudaSoA<floatType> cell1,
		LJFunctorCudaSoA<floatType> cell2, hipStream_t stream) {
	switch (_num_threads) {
	case 32:
		SoAFunctorN3Pair<floatType, 32> <<<numRequiredBlocks(cell1._size), 32,
				0, stream>>>(cell1, cell2);
		break;
	case 64:
		SoAFunctorN3Pair<floatType, 64> <<<numRequiredBlocks(cell1._size), 64,
				0, stream>>>(cell1, cell2);
		break;
	case 128:
		SoAFunctorN3Pair<floatType, 128> <<<numRequiredBlocks(cell1._size), 128,
				0, stream>>>(cell1, cell2);
		break;
	case 256:
		SoAFunctorN3Pair<floatType, 256> <<<numRequiredBlocks(cell1._size), 256,
				0, stream>>>(cell1, cell2);
		break;
	case 512:
		SoAFunctorN3Pair<floatType, 512> <<<numRequiredBlocks(cell1._size), 512,
				0, stream>>>(cell1, cell2);
		break;
	case 1024:
		SoAFunctorN3Pair<floatType, 1024> <<<numRequiredBlocks(cell1._size),
				1024, 0, stream>>>(cell1, cell2);
		break;
	default:
		autopas::utils::ExceptionHandler::exception(
				std::string("cuda Kernel size not available"));
		break;
	}
	autopas::utils::CudaExceptionHandler::checkLastCudaCall();

}

template void CudaWrapper::SoAFunctorN3PairWrapper<float>(
		LJFunctorCudaSoA<float> cell1, LJFunctorCudaSoA<float> cell2,
		hipStream_t stream);
template void CudaWrapper::SoAFunctorN3PairWrapper<double>(
		LJFunctorCudaSoA<double> cell1, LJFunctorCudaSoA<double> cell2,
		hipStream_t stream);

template<typename floatType, int block_size>
__global__
void LinkedCellsTraversalNoN3(LJFunctorCudaSoA<floatType> cell,
		unsigned int* cids, size_t* cellSizes, unsigned int offsets_size,
		int* offsets) {

	int own_cid = cids[blockIdx.x];
	__shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
	typename vec3<floatType>::Type myposition = { getInfinity<floatType>(),
			getInfinity<floatType>(), getInfinity<floatType>() };
	typename vec3<floatType>::Type myf = { 0, 0, 0 };

	int index = cellSizes[own_cid] + threadIdx.x;
	if (threadIdx.x < (cellSizes[own_cid + 1] - cellSizes[own_cid])) {
		myposition.x = cell._posX[index];
		myposition.y = cell._posY[index];
		myposition.z = cell._posZ[index];
	}
	//other cells
	for (auto other_index = 0; other_index < offsets_size; ++other_index) {
		const int other_id = own_cid + offsets[other_index];
		const int cell2Start = cellSizes[other_id];
		const int sizeCell2 = cellSizes[other_id + 1] - cell2Start;

		cell2_pos_shared[threadIdx.x] = {cell._posX[cell2Start+threadIdx.x], cell._posY[cell2Start+threadIdx.x], cell._posZ[cell2Start+threadIdx.x]};
		__syncthreads();
		for (int j = 0; j < sizeCell2; ++j) {
			myf = bodyBodyF<floatType>(myposition, cell2_pos_shared[j], myf);
		}
		__syncthreads();
	}
	if (threadIdx.x < (cellSizes[own_cid + 1] - cellSizes[own_cid])) {
		atomicAdd(cell._forceX + index, myf.x);
		atomicAdd(cell._forceY + index, myf.y);
		atomicAdd(cell._forceZ + index, myf.z);
	}
}

template<typename floatType, int block_size>
__global__
void LinkedCellsTraversalN3(LJFunctorCudaSoA<floatType> cell,
		unsigned int* cids, size_t* cellSizes, unsigned int offsets_size,
		int* offsets) {

	int own_cid = cids[blockIdx.x];
	__shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
	__shared__ typename vec3<floatType>::Type cell2_forces_shared[block_size];

	typename vec3<floatType>::Type myposition = { getInfinity<floatType>(),
			getInfinity<floatType>(), getInfinity<floatType>() };
	typename vec3<floatType>::Type myf = { 0, 0, 0 };

	int index = cellSizes[own_cid] + threadIdx.x;
	if (threadIdx.x < (cellSizes[own_cid + 1] - cellSizes[own_cid])) {
		myposition.x = cell._posX[index];
		myposition.y = cell._posY[index];
		myposition.z = cell._posZ[index];
	}
	//other cells
	for (auto other_index = 0; other_index < offsets_size; ++other_index) {
		const int other_id = own_cid + offsets[other_index];
		const int cell2Start = cellSizes[other_id];
		const int sizeCell2 = cellSizes[other_id + 1] - cell2Start;

		cell2_pos_shared[threadIdx.x] = {cell._posX[cell2Start+threadIdx.x], cell._posY[cell2Start+threadIdx.x], cell._posZ[cell2Start+threadIdx.x]};
		cell2_forces_shared[threadIdx.x] = {0,0,0};
		__syncthreads();
		for (int j = 0; j < sizeCell2; ++j) {
			const int offset = (j + threadIdx.x) % sizeCell2;
			myf = bodyBodyFN3<floatType, false>(myposition,
					cell2_pos_shared[offset], myf,
					cell2_forces_shared + offset);
		}
		__syncthreads();

		atomicAdd(cell._forceX + cell2Start + threadIdx.x,
				cell2_forces_shared[threadIdx.x].x);
		atomicAdd(cell._forceY + cell2Start + threadIdx.x,
				cell2_forces_shared[threadIdx.x].y);
		atomicAdd(cell._forceZ + cell2Start + threadIdx.x,
				cell2_forces_shared[threadIdx.x].z);
		__syncthreads();
	}
	//same cells
	{
		const int cell1Start = cellSizes[own_cid];
		const int sizeCell1 = cellSizes[own_cid + 1] - cell1Start;

		cell2_pos_shared[threadIdx.x] = {cell._posX[cell1Start+threadIdx.x], cell._posY[cell1Start+threadIdx.x], cell._posZ[cell1Start+threadIdx.x]};
		__syncthreads();
		for (int j = 0; j < sizeCell1; ++j) {
			myf = bodyBodyF<floatType>(myposition, cell2_pos_shared[j], myf);
		}
		__syncthreads();
	}
	if (threadIdx.x < (cellSizes[own_cid + 1] - cellSizes[own_cid])) {
		atomicAdd(cell._forceX + index, myf.x);
		atomicAdd(cell._forceY + index, myf.y);
		atomicAdd(cell._forceZ + index, myf.z);
	}
}

template<typename floatType>
void CudaWrapper::LinkedCellsTraversalNoN3Wrapper(
		LJFunctorCudaSoA<floatType> cell1, unsigned int cids_size,
		unsigned int* cids, unsigned int cellSizes_size, size_t* cellSizes,
		unsigned int offsets_size, int* offsets, hipStream_t stream) {
	switch (_num_threads) {
	case 32:
		LinkedCellsTraversalNoN3<floatType, 32> <<<cids_size, 32, 0, stream>>>(
				cell1, cids, cellSizes, offsets_size, offsets);
		break;
	case 64:
		LinkedCellsTraversalNoN3<floatType, 64> <<<cids_size, 64, 0, stream>>>(
				cell1, cids, cellSizes, offsets_size, offsets);
		break;
	case 96:
		LinkedCellsTraversalNoN3<floatType, 96> <<<cids_size, 96, 0, stream>>>(
				cell1, cids, cellSizes, offsets_size, offsets);
		break;
	default:
		autopas::utils::ExceptionHandler::exception(
				"cuda Kernel size not available for Linked cells available 32, 64, 96. Too many particles in a cell. Requested: {}",
				_num_threads);
		break;
	}
	autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template void CudaWrapper::LinkedCellsTraversalNoN3Wrapper<float>(
		LJFunctorCudaSoA<float> cell1, unsigned int cids_size,
		unsigned int* cids, unsigned int cellSizes_size, size_t* cellSizes,
		unsigned int offsets_size, int* offsets, hipStream_t stream);

template void CudaWrapper::LinkedCellsTraversalNoN3Wrapper<double>(
		LJFunctorCudaSoA<double> cell1, unsigned int cids_size,
		unsigned int* cids, unsigned int cellSizes_size, size_t* cellSizes,
		unsigned int offsets_size, int* offsets, hipStream_t stream);

template<typename floatType>
void CudaWrapper::LinkedCellsTraversalN3Wrapper(
		LJFunctorCudaSoA<floatType> cell1, unsigned int cids_size,
		unsigned int* cids, unsigned int cellSizes_size, size_t* cellSizes,
		unsigned int offsets_size, int* offsets, hipStream_t stream) {
	switch (_num_threads) {
	case 32:
		LinkedCellsTraversalN3<floatType, 32> <<<cids_size, 32, 0, stream>>>(
				cell1, cids, cellSizes, offsets_size, offsets);
		break;
	case 64:
		LinkedCellsTraversalN3<floatType, 64> <<<cids_size, 64, 0, stream>>>(
				cell1, cids, cellSizes, offsets_size, offsets);
		break;
	case 96:
		LinkedCellsTraversalN3<floatType, 96> <<<cids_size, 96, 0, stream>>>(
				cell1, cids, cellSizes, offsets_size, offsets);
		break;
	default:
		autopas::utils::ExceptionHandler::exception(
				"cuda Kernel size not available for Linked cells available 32, 64, 96. Too many particles in a cell. Requested: {}",
				_num_threads);
		break;
	}
	autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template void CudaWrapper::LinkedCellsTraversalN3Wrapper<float>(
		LJFunctorCudaSoA<float> cell1, unsigned int cids_size,
		unsigned int* cids, unsigned int cellSizes_size, size_t* cellSizes,
		unsigned int offsets_size, int* offsets, hipStream_t stream);

template void CudaWrapper::LinkedCellsTraversalN3Wrapper<double>(
		LJFunctorCudaSoA<double> cell1, unsigned int cids_size,
		unsigned int* cids, unsigned int cellSizes_size, size_t* cellSizes,
		unsigned int offsets_size, int* offsets, hipStream_t stream);

template<typename floatType, int block_size>
__global__
void CellVerletTraversalNoN3(LJFunctorCudaSoA<floatType> cell,
		unsigned int others_size, unsigned int* other_ids) {

	__shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
	typename vec3<floatType>::Type myposition = { getInfinity<floatType>(),
			getInfinity<floatType>(), getInfinity<floatType>() };
	typename vec3<floatType>::Type myf = { 0, 0, 0 };

	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	myposition.x = cell._posX[index];
	myposition.y = cell._posY[index];
	myposition.z = cell._posZ[index];

	//other cells
	for (auto other_index = others_size * blockIdx.x;
			other_ids[other_index] < UINT_MAX; ++other_index) {
		const unsigned int cell2Start = blockDim.x * other_ids[other_index];

		cell2_pos_shared[threadIdx.x] = {cell._posX[cell2Start+threadIdx.x], cell._posY[cell2Start+threadIdx.x], cell._posZ[cell2Start+threadIdx.x]};
		__syncthreads();
		for (int j = 0; j < block_size; ++j) {
			myf = bodyBodyF<floatType>(myposition, cell2_pos_shared[j], myf);
		}
		__syncthreads();
	}
	atomicAdd(cell._forceX + index, myf.x);
	atomicAdd(cell._forceY + index, myf.y);
	atomicAdd(cell._forceZ + index, myf.z);
}

template<typename floatType>
void CudaWrapper::CellVerletTraversalNoN3Wrapper(
		LJFunctorCudaSoA<floatType> cell1, unsigned int ncells,
		unsigned int clusterSize, unsigned int others_size,
		unsigned int* other_ids, hipStream_t stream) {
	switch (clusterSize) {
	case 32:
		CellVerletTraversalNoN3<floatType, 32> <<<ncells, 32, 0, stream>>>(
				cell1, others_size, other_ids);
		break;
	case 64:
		CellVerletTraversalNoN3<floatType, 64> <<<ncells, 64, 0, stream>>>(
				cell1, others_size, other_ids);
		break;
	case 96:
		CellVerletTraversalNoN3<floatType, 96> <<<ncells, 96, 0, stream>>>(
				cell1, others_size, other_ids);
		break;
	default:
		autopas::utils::ExceptionHandler::exception(
				"cuda Kernel size not available for Verlet cells available 32, 64, 96. Too many particles in a cell. Requested: {}",
				_num_threads);
		break;
	}
	autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template void CudaWrapper::CellVerletTraversalNoN3Wrapper<float>(
		LJFunctorCudaSoA<float> cell1, unsigned int ncells,
		unsigned int clusterSize, unsigned int others_size,
		unsigned int* other_ids, hipStream_t stream);

template void CudaWrapper::CellVerletTraversalNoN3Wrapper<double>(
		LJFunctorCudaSoA<double> cell1, unsigned int ncells,
		unsigned int clusterSize, unsigned int others_size,
		unsigned int* other_ids, hipStream_t stream);

template<typename floatType, int block_size>
__global__
void CellVerletTraversalN3(LJFunctorCudaSoA<floatType> cell,
		unsigned int others_size, unsigned int* other_ids) {
	const unsigned int mask = block_size - 1;

	__shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
	__shared__ typename vec3<floatType>::Type cell2_forces_shared[block_size];

	typename vec3<floatType>::Type myposition = { getInfinity<floatType>(),
			getInfinity<floatType>(), getInfinity<floatType>() };
	typename vec3<floatType>::Type myf = { 0, 0, 0 };

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	myposition.x = cell._posX[index];
	myposition.y = cell._posY[index];
	myposition.z = cell._posZ[index];

	//other cells
	for (auto other_index = others_size * blockIdx.x;
			other_ids[other_index] != UINT_MAX; ++other_index) {
		const unsigned int cell2Start = blockDim.x * other_ids[other_index];

		cell2_pos_shared[threadIdx.x] = {cell._posX[cell2Start+threadIdx.x], cell._posY[cell2Start+threadIdx.x], cell._posZ[cell2Start+threadIdx.x]};
		cell2_forces_shared[threadIdx.x] = {0,0,0};
		__syncthreads();
		for (int j = 0; j < block_size; ++j) {
			unsigned int offset = 0;
			if ((block_size & (block_size - 1)) == 0) {
				offset = (j + threadIdx.x) & mask;
			} else {
				offset = (j + threadIdx.x) % block_size;
			}
			myf = bodyBodyFN3<floatType, false>(myposition,
					cell2_pos_shared[offset], myf,
					cell2_forces_shared + offset);
		}
		__syncthreads();

		atomicAdd(cell._forceX + cell2Start + threadIdx.x,
				cell2_forces_shared[threadIdx.x].x);
		atomicAdd(cell._forceY + cell2Start + threadIdx.x,
				cell2_forces_shared[threadIdx.x].y);
		atomicAdd(cell._forceZ + cell2Start + threadIdx.x,
				cell2_forces_shared[threadIdx.x].z);
		__syncthreads();
	}

	//same cluster without N3
	{
		const unsigned int cellStart = blockIdx.x * blockDim.x;

		cell2_pos_shared[threadIdx.x] = {cell._posX[cellStart+threadIdx.x], cell._posY[cellStart+threadIdx.x], cell._posZ[cellStart+threadIdx.x]};
		__syncthreads();
		for (int j = 0; j < block_size; ++j) {
			myf = bodyBodyF<floatType>(myposition, cell2_pos_shared[j], myf);
		}
		__syncthreads();
	}
	atomicAdd(cell._forceX + index, myf.x);
	atomicAdd(cell._forceY + index, myf.y);
	atomicAdd(cell._forceZ + index, myf.z);
}

template<typename floatType>
void CudaWrapper::CellVerletTraversalN3Wrapper(
		LJFunctorCudaSoA<floatType> cell1, unsigned int ncells,
		unsigned int clusterSize, unsigned int others_size,
		unsigned int* other_ids, hipStream_t stream) {
	switch (clusterSize) {
	case 32:
		CellVerletTraversalN3<floatType, 32> <<<ncells, 32, 0, stream>>>(cell1,
				others_size, other_ids);
		break;
	case 64:
		CellVerletTraversalN3<floatType, 64> <<<ncells, 64, 0, stream>>>(cell1,
				others_size, other_ids);
		break;
	case 96:
		CellVerletTraversalN3<floatType, 96> <<<ncells, 96, 0, stream>>>(cell1,
				others_size, other_ids);
		break;
	default:
		autopas::utils::ExceptionHandler::exception(
				"cuda Kernel size not available for Verlet cells available 32, 64, 96. Too many particles in a cell. Requested: {}",
				_num_threads);
		break;
	}
	autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template void CudaWrapper::CellVerletTraversalN3Wrapper<float>(
		LJFunctorCudaSoA<float> cell1, unsigned int ncells,
		unsigned int clusterSize, unsigned int others_size,
		unsigned int* other_ids, hipStream_t stream);

template void CudaWrapper::CellVerletTraversalN3Wrapper<double>(
		LJFunctorCudaSoA<double> cell1, unsigned int ncells,
		unsigned int clusterSize, unsigned int others_size,
		unsigned int* other_ids, hipStream_t stream);

template<typename floatType>
void CudaWrapper::loadConstants(floatType cutoffsquare, floatType epsilon24,
		floatType sigmasquare) {

	constants<floatType> c;
	c.cutoffsquare = cutoffsquare;
	c.epsilon24 = epsilon24;
	c.sigmasquare = sigmasquare;
	hipMemcpyToSymbol(HIP_SYMBOL(global_constants_double), &c,
			sizeof(constants<floatType> ));
}
template void CudaWrapper::loadConstants<float>(float cutoffsquare,
		float epsilon24, float sigmasquare);
template void CudaWrapper::loadConstants<double>(double cutoffsquare,
		double epsilon24, double sigmasquare);

} // namespace autopas
