#include "hip/hip_runtime.h"
/**
 * @file LJFunctorCuda.h
 *
 * @date 9.1.2019
 * @author jspahl
 */
#include "autopas/pairwiseFunctors/LJFunctorCuda.cuh"

__constant__ constants global_constants;

__device__
double3 bodyBodyF(double3 i,
		double3 j, double3 fi) {
  double drx = i.x - j.x;
  double dry = i.y - j.y;
  double drz = i.z - j.z;

  double dr2 = drx* drx + dry * dry + drz* drz;

  if (dr2 > global_constants.cutoffsquare | dr2 == 0.0) {
  	return fi;
  }

  double invdr2 = 1. / dr2;
  double lj6 = global_constants.sigmasquare * invdr2;
  lj6 = lj6 * lj6 * lj6;
  double lj12 = lj6 * lj6;
  double lj12m6 = lj12 - lj6;
  double fac = global_constants.epsilon24 * (lj12 + lj12m6) * invdr2;

  fi.x += drx * fac;
  fi.y += dry * fac;
  fi.z += drz * fac;

  return fi;
}


template <int block_size>
__global__
void SoAFunctorNoN3(int N, double* posX, double* posY, double* posZ, double* forceX, double* forceY, double* forceZ){
	 __shared__ double3 block_pos[block_size];
	 int i, tile;
	 int tid = blockIdx.x * blockDim.x + threadIdx.x;
	 if(tid >= N){
		 return;
	 }

	 double3 myposition;
	 myposition.x = posX[tid];
	 myposition.y = posY[tid];
	 myposition.z = posZ[tid];
	 double3 myf = {forceX[tid], forceY[tid], forceZ[tid]};

	 for(i = 0, tile = 0; i < N; i+=block_size, ++tile){
		 int idx = tile * blockDim.x + threadIdx.x;

		 block_pos[threadIdx.x] = {posX[idx], posY[idx], posZ[idx]};
		 __syncthreads();

		  for(int j = 0; j < blockDim.x; ++j){
			  myf = bodyBodyF(myposition, block_pos[j], myf);
		  }
		 __syncthreads();
	 }

	 forceX[tid] = myf.x;
	 forceY[tid] = myf.y;
	 forceZ[tid] = myf.z;
}

template <int block_size>
__global__
void SoAFunctorNoN3Pair(int N, double* posX, double* posY, double* posZ, double* forceX, double* forceY, double* forceZ,
		int M, double* posX2, double* posY2, double* posZ2){
	 __shared__ double3 block_pos[block_size];
	 int i, tile;
	 int tid = blockIdx.x * blockDim.x + threadIdx.x;
	 if(tid >= N){
		 return;
	 }

	 double3 myposition;
	 myposition.x = posX[tid];
	 myposition.y = posY[tid];
	 myposition.z = posZ[tid];
	 double3 myf = {forceX[tid], forceY[tid], forceZ[tid]};

	 for(i = 0, tile = 0; i < M; i+=block_size, ++tile){
		 int idx = tile * blockDim.x + threadIdx.x;

		 block_pos[threadIdx.x] = {posX2[idx], posY2[idx], posZ2[idx]};
		 __syncthreads();

		  for(int j = 0; j < blockDim.x; ++j){
			  myf = bodyBodyF(myposition, block_pos[j], myf);
		  }
		 __syncthreads();
	 }

	 forceX[tid] = myf.x;
	 forceY[tid] = myf.y;
	 forceZ[tid] = myf.z;
}

template <int block_size>
__global__
void AoSFunctorNoN3(int N, double* particles){
	 __shared__ double3 block_pos[block_size];
	 int i, tile;
	 int tid = blockIdx.x * blockDim.x + threadIdx.x;
	 if(tid >= N){
		 return;
	 }

	 double3 myposition;
	 myposition.x = particles[6 * tid + 0];
	 myposition.y = particles[6 * tid + 1];
	 myposition.z = particles[6 * tid + 2];
	 double3 myf = {particles[6 * tid + 3],particles[6 * tid + 4],particles[6 * tid + 5]};

	 for(i = 0, tile = 0; i < N; i+=block_size, ++tile){
		 int idx = tile * blockDim.x + threadIdx.x;

		 block_pos[threadIdx.x] = {particles[6 * idx + 0], particles[6 * idx + 1], particles[6 * idx + 2]};
		 __syncthreads();

		  for(int j = 0; j < blockDim.x; ++j){
			  myf = bodyBodyF(myposition, block_pos[j], myf);
		  }
		 __syncthreads();
	 }

	 particles[6 * tid + 3] = myf.x;
	 particles[6 * tid + 4] = myf.y;
	 particles[6 * tid + 5] = myf.z;
}

template <int block_size>
__global__
void AoSFunctorNoN3Pair(int N, int M, double* particles1, double* particles2){
	 int i, tile;
	 int tid = blockIdx.x * blockDim.x + threadIdx.x;
	 __shared__ double3 block_pos2[32];

	 double3 myposition;
	 myposition.x = particles1[6 * tid + 0];
	 myposition.y = particles1[6 * tid + 1];
	 myposition.z = particles1[6 * tid + 2];
	 double3 myf = {0,0,0 };

	 for(i = 0, tile = 0; i < M; i+=block_size, ++tile){
		 int idx = tile * blockDim.x + threadIdx.x;

		 block_pos2[threadIdx.x] = {particles2[6 * idx + 0], particles2[6 * idx + 1], particles2[6 * idx + 2]};
		 __syncthreads();

		  for(int j = 0; j < blockDim.x; ++j){
			  myf = bodyBodyF(myposition, block_pos2[j], myf);
		  }
		 __syncthreads();
	 }

	 particles1[6 * tid + 3] = myf.x;
	 particles1[6 * tid + 4] = myf.y;
	 particles1[6 * tid + 5] = myf.z;
}

void AoSFunctorNoN3Wrapper(int N, double* particles){
	AoSFunctorNoN3<32><<<N/32 + 1,32>>>(N, particles);
}

void AoSFunctorNoN3PairWrapper(int N, int M, double* particles1, double* particles2){
	AoSFunctorNoN3Pair<32><<<N/32 + 1,32>>>(N, M, particles1, particles2);
}

void SoAFunctorNoN3Wrapper(int N, double* posX, double* posY, double* posZ, double* forceX, double* forceY, double* forceZ){
	SoAFunctorNoN3<32><<<N/32 + 1,32>>>(N, posX, posY, posZ, forceX, forceY, forceZ);
}

void SoAFunctorNoN3PairWrapper(int N, double* posX, double* posY, double* posZ, double* forceX, double* forceY, double* forceZ,
		int M, double* posX2, double* posY2, double* posZ2){
	SoAFunctorNoN3Pair<32><<<N/32 + 1,32>>>(N, posX, posY, posZ, forceX, forceY, forceZ, M, posX2, posY2, posZ2);
}

void loadConstants(double cutoffsquare, double epsilon24, double sigmasquare){

	constants c;
	c.cutoffsquare = cutoffsquare;
	c.epsilon24 = epsilon24;
	c.sigmasquare = sigmasquare;

	hipMemcpyToSymbol(HIP_SYMBOL(global_constants), &c, sizeof(constants));
}

//template __global__ void SoAFunctorNoN3<32>(int N, double* posX, double* posY, double* posZ, double* forceX, double* forceY, double* forceZ);

