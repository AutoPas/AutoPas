#include "hip/hip_runtime.h"
/**
 * @file LJFunctorCuda.h
 *
 * @date 9.1.2019
 * @author jspahl
 */
#include <iostream>
#include "autopas/pairwiseFunctors/LJFunctorCuda.cuh"
#include "autopas/utils/CudaExceptionHandler.h"
#include "autopas/utils/ExceptionHandler.h"
#include "hip/hip_math_constants.h"

namespace autopas {

__constant__ LJFunctorConstants<float> global_constants_float;
__constant__ LJFunctorConstants<double> global_constants_double;

template <typename T>
__device__ inline LJFunctorConstants<T> &getConstants() {
  return global_constants_float;
}
template <>
__device__ inline LJFunctorConstants<double> &getConstants<double>() {
  return global_constants_double;
}

template <typename T>
__device__ inline T getInfinity() {
  return HIP_INF_F;
}
template <>
__device__ inline double getInfinity<double>() {
  return HIP_INF;
}

template <typename floatType>
__device__ inline typename vec3<floatType>::Type bodyBodyF(typename vec3<floatType>::Type i,
                                                           typename vec3<floatType>::Type j,
                                                           typename vec3<floatType>::Type fi) {
  floatType drx = i.x - j.x;
  floatType dry = i.y - j.y;
  floatType drz = i.z - j.z;

  floatType dr2 = drx * drx + dry * dry + drz * drz;

  if (dr2 > getConstants<floatType>().cutoffsquare | dr2 == 0.0) {
    return fi;
  }

  floatType invdr2 = 1. / dr2;
  floatType lj6 = getConstants<floatType>().sigmasquare * invdr2;
  lj6 = lj6 * lj6 * lj6;
  floatType lj12 = lj6 * lj6;
  floatType lj12m6 = lj12 - lj6;
  floatType fac = getConstants<floatType>().epsilon24 * (lj12 + lj12m6) * invdr2;

  fi.x += drx * fac;
  fi.y += dry * fac;
  fi.z += drz * fac;

  return fi;
}

template <typename floatType, bool n3AdditionSafe = false>
__device__ inline typename vec3<floatType>::Type bodyBodyFN3(typename vec3<floatType>::Type i,
                                                             typename vec3<floatType>::Type j,
                                                             typename vec3<floatType>::Type fi,
                                                             typename vec3<floatType>::Type *fj) {
  floatType drx = i.x - j.x;
  floatType dry = i.y - j.y;
  floatType drz = i.z - j.z;

  floatType dr2 = drx * drx + dry * dry + drz * drz;

  if (dr2 > getConstants<floatType>().cutoffsquare) {
    return fi;
  }

  floatType invdr2 = 1. / dr2;
  floatType lj6 = getConstants<floatType>().sigmasquare * invdr2;
  lj6 = lj6 * lj6 * lj6;
  floatType lj12 = lj6 * lj6;
  floatType lj12m6 = lj12 - lj6;
  floatType fac = getConstants<floatType>().epsilon24 * (lj12 + lj12m6) * invdr2;

  floatType dfx = drx * fac;
  floatType dfy = dry * fac;
  floatType dfz = drz * fac;

  fi.x += dfx;
  fi.y += dfy;
  fi.z += dfz;

  if (n3AdditionSafe) {
    fj->x -= dfx;
    fj->y -= dfy;
    fj->z -= dfz;
  } else {
    atomicAdd(&(fj->x), -dfx);
    atomicAdd(&(fj->y), -dfy);
    atomicAdd(&(fj->z), -dfz);
  }
  return fi;
}

template <typename floatType, int block_size>
__global__ void SoAFunctorNoN3(LJFunctorCudaSoA<floatType> cell1) {
  __shared__ typename vec3<floatType>::Type block_pos[block_size];
  int i, tile;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  typename vec3<floatType>::Type myposition = {0, 0, 0};
  typename vec3<floatType>::Type myf = {0, 0, 0};
  if (tid < cell1._size) {
    myposition.x = cell1._posX[tid];
    myposition.y = cell1._posY[tid];
    myposition.z = cell1._posZ[tid];
  }

  for (i = block_size, tile = 0; i < cell1._size; i += block_size, ++tile) {
    int idx = tile * block_size + threadIdx.x;

    block_pos[threadIdx.x] = {cell1._posX[idx], cell1._posY[idx], cell1._posZ[idx]};
    __syncthreads();
    if (tid < cell1._size) {
      for (int j = 0; j < blockDim.x; ++j) {
        myf = bodyBodyF<floatType>(myposition, block_pos[j], myf);
      }
    }
    __syncthreads();
  }
  {
    int idx = tile * block_size + threadIdx.x;
    block_pos[threadIdx.x] = {cell1._posX[idx], cell1._posY[idx], cell1._posZ[idx]};
    __syncthreads();

    const int size = cell1._size - tile * blockDim.x;
    for (int j = 0; j < size; ++j) {
      myf = bodyBodyF<floatType>(myposition, block_pos[j], myf);
    }

    __syncthreads();
  }

  atomicAdd(cell1._forceX + tid, myf.x);
  atomicAdd(cell1._forceY + tid, myf.y);
  atomicAdd(cell1._forceZ + tid, myf.z);
}

template <typename floatType, int block_size>
__global__ void SoAFunctorNoN3Pair(LJFunctorCudaSoA<floatType> cell1, LJFunctorCudaSoA<floatType> cell2) {
  __shared__ typename vec3<floatType>::Type block_pos[block_size];
  int i, tile;
  int tid = blockIdx.x * block_size + threadIdx.x;
  typename vec3<floatType>::Type myposition;
  typename vec3<floatType>::Type myf = {0, 0, 0};

  if (tid < cell1._size) {
    myposition.x = cell1._posX[tid];
    myposition.y = cell1._posY[tid];
    myposition.z = cell1._posZ[tid];
  }

  for (i = 0, tile = 0; i < cell2._size; i += block_size, ++tile) {
    int idx = tile * block_size + threadIdx.x;

    if (idx < cell2._size) block_pos[threadIdx.x] = {cell2._posX[idx], cell2._posY[idx], cell2._posZ[idx]};
    __syncthreads();

    const int size = min(block_size, cell2._size - i);
    for (int j = 0; j < size; ++j) {
      myf = bodyBodyF<floatType>(myposition, block_pos[j], myf);
    }
    __syncthreads();
  }
  atomicAdd(cell1._forceX + tid, myf.x);
  atomicAdd(cell1._forceY + tid, myf.y);
  atomicAdd(cell1._forceZ + tid, myf.z);
}

template <typename floatType, int block_size, bool NMisMultipleBlockSize = false>
__global__ void SoAFunctorN3(LJFunctorCudaSoA<floatType> cell1) {
  static_assert((block_size & (block_size - 1)) == 0, "block size must be power of 2");
  __shared__ typename vec3<floatType>::Type cell1_pos_shared[block_size];
  __shared__ typename vec3<floatType>::Type cell1_forces_shared[block_size];
  int tid = blockIdx.x * block_size + threadIdx.x;
  typename vec3<floatType>::Type myposition = {getInfinity<floatType>(), getInfinity<floatType>(),
                                               getInfinity<floatType>()};
  typename vec3<floatType>::Type myf = {0, 0, 0};
  int i, tile;
  const int mask = block_size - 1;

  if (not NMisMultipleBlockSize && tid < cell1._size) {
    myposition.x = cell1._posX[tid];
    myposition.y = cell1._posY[tid];
    myposition.z = cell1._posZ[tid];
  }

  for (i = 0, tile = 0; tile < blockIdx.x; i += block_size, ++tile) {
    int idx = tile * block_size + threadIdx.x;
    cell1_pos_shared[threadIdx.x] = {cell1._posX[idx], cell1._posY[idx], cell1._posZ[idx]};
    cell1_forces_shared[threadIdx.x] = {0, 0, 0};
    __syncthreads();

    for (int j = 0; j < block_size; ++j) {
      const int offset = (j + threadIdx.x) & mask;
      myf = bodyBodyFN3<floatType>(myposition, cell1_pos_shared[offset], myf, cell1_forces_shared + offset);
    }
    __syncthreads();

    atomicAdd(cell1._forceX + idx, cell1_forces_shared[threadIdx.x].x);
    atomicAdd(cell1._forceY + idx, cell1_forces_shared[threadIdx.x].y);
    atomicAdd(cell1._forceZ + idx, cell1_forces_shared[threadIdx.x].z);
    __syncthreads();
  }

  {
    int idx = blockIdx.x * block_size + threadIdx.x;
    cell1_pos_shared[threadIdx.x] = {cell1._posX[idx], cell1._posY[idx], cell1._posZ[idx]};
    cell1_forces_shared[threadIdx.x] = {0, 0, 0};
    __syncthreads();

    for (int j = threadIdx.x - 1; j >= 0; --j) {
      myf = bodyBodyFN3<floatType>(myposition, cell1_pos_shared[j], myf, cell1_forces_shared + j);
    }
    __syncthreads();

    atomicAdd(cell1._forceX + idx, cell1_forces_shared[threadIdx.x].x);
    atomicAdd(cell1._forceY + idx, cell1_forces_shared[threadIdx.x].y);
    atomicAdd(cell1._forceZ + idx, cell1_forces_shared[threadIdx.x].z);
    __syncthreads();
  }

  atomicAdd(cell1._forceX + tid, myf.x);
  atomicAdd(cell1._forceY + tid, myf.y);
  atomicAdd(cell1._forceZ + tid, myf.z);
}

template <typename floatType, int block_size, bool NMisMultipleBlockSize = false>
__global__ void SoAFunctorN3Pair(LJFunctorCudaSoA<floatType> cell1, LJFunctorCudaSoA<floatType> cell2) {
  static_assert((block_size & (block_size - 1)) == 0, "block size must be power of 2");
  __shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
  __shared__ typename vec3<floatType>::Type cell2_forces_shared[block_size];
  int tid = blockIdx.x * block_size + threadIdx.x;
  typename vec3<floatType>::Type myposition = {getInfinity<floatType>(), getInfinity<floatType>(),
                                               getInfinity<floatType>()};
  typename vec3<floatType>::Type myf = {0, 0, 0};
  int i, tile;
  const int mask = block_size - 1;

  if (not NMisMultipleBlockSize && tid < cell1._size) {
    myposition.x = cell1._posX[tid];
    myposition.y = cell1._posY[tid];
    myposition.z = cell1._posZ[tid];
  }
  for (i = block_size, tile = 0; i <= cell2._size; i += block_size, ++tile) {
    int idx = tile * block_size + threadIdx.x;
    cell2_pos_shared[threadIdx.x] = {cell2._posX[idx], cell2._posY[idx], cell2._posZ[idx]};
    cell2_forces_shared[threadIdx.x] = {0, 0, 0};
    __syncthreads();

    for (int j = 0; j < block_size; ++j) {
      const int offset = (j + threadIdx.x) & mask;
      myf = bodyBodyFN3<floatType, true>(myposition, cell2_pos_shared[offset], myf, cell2_forces_shared + offset);
    }
    __syncthreads();

    atomicAdd(cell2._forceX + idx, cell2_forces_shared[threadIdx.x].x);
    atomicAdd(cell2._forceY + idx, cell2_forces_shared[threadIdx.x].y);
    atomicAdd(cell2._forceZ + idx, cell2_forces_shared[threadIdx.x].z);
    __syncthreads();
  }
  if ((not NMisMultipleBlockSize) && (i > cell2._size)) {
    int idx = tile * block_size + threadIdx.x;
    if (idx < cell2._size) {
      cell2_pos_shared[threadIdx.x] = {cell2._posX[idx], cell2._posY[idx], cell2._posZ[idx]};
      cell2_forces_shared[threadIdx.x] = {0, 0, 0};
    }
    __syncthreads();

    const int size = block_size + cell2._size - i;
    for (int j = 0; j < size; ++j) {
      const int offset = (j + threadIdx.x) % size;
      myf = bodyBodyFN3<floatType>(myposition, cell2_pos_shared[offset], myf, cell2_forces_shared + offset);
    }
    __syncthreads();
    if (idx < cell2._size) {
      atomicAdd(cell2._forceX + idx, cell2_forces_shared[threadIdx.x].x);
      atomicAdd(cell2._forceY + idx, cell2_forces_shared[threadIdx.x].y);
      atomicAdd(cell2._forceZ + idx, cell2_forces_shared[threadIdx.x].z);
      __syncthreads();
    }
  }

  atomicAdd(cell1._forceX + tid, myf.x);
  atomicAdd(cell1._forceY + tid, myf.y);
  atomicAdd(cell1._forceZ + tid, myf.z);
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::SoAFunctorNoN3Wrapper(FunctorCudaSoA<floatType> *cell1Base, hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType> *>(cell1Base);

  switch (_num_threads) {
    case 32:
      SoAFunctorNoN3<floatType, 32><<<numRequiredBlocks(cell1._size), 32, 0, stream>>>(cell1);
      break;
    case 64:
      SoAFunctorNoN3<floatType, 64><<<numRequiredBlocks(cell1._size), 64, 0, stream>>>(cell1);
      break;
    case 96:
      SoAFunctorNoN3<floatType, 96><<<numRequiredBlocks(cell1._size), 96, 0, stream>>>(cell1);
      break;
    case 128:
      SoAFunctorNoN3<floatType, 128><<<numRequiredBlocks(cell1._size), 128, 0, stream>>>(cell1);
      break;
    case 256:
      SoAFunctorNoN3<floatType, 256><<<numRequiredBlocks(cell1._size), 256, 0, stream>>>(cell1);
      break;
    case 512:
      SoAFunctorNoN3<floatType, 512><<<numRequiredBlocks(cell1._size), 512, 0, stream>>>(cell1);
      break;
    case 1024:
      SoAFunctorNoN3<floatType, 1024><<<numRequiredBlocks(cell1._size), 1024, 0, stream>>>(cell1);
      break;
    default:
      autopas::utils::ExceptionHandler::exception(std::string("cuda Kernel size not available"));
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::SoAFunctorNoN3PairWrapper(FunctorCudaSoA<floatType> *cell1Base,
                                                                FunctorCudaSoA<floatType> *cell2Base,
                                                                hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType> *>(cell1Base);
  LJFunctorCudaSoA<floatType> cell2 = *static_cast<LJFunctorCudaSoA<floatType> *>(cell2Base);

  switch (_num_threads) {
    case 32:
      SoAFunctorNoN3Pair<floatType, 32><<<numRequiredBlocks(cell1._size), 32, 0, stream>>>(cell1, cell2);
      break;
    case 64:
      SoAFunctorNoN3Pair<floatType, 64><<<numRequiredBlocks(cell1._size), 64, 0, stream>>>(cell1, cell2);
      break;
    case 96:
      SoAFunctorNoN3Pair<floatType, 96><<<numRequiredBlocks(cell1._size), 96, 0, stream>>>(cell1, cell2);
      break;
    case 128:
      SoAFunctorNoN3Pair<floatType, 128><<<numRequiredBlocks(cell1._size), 128, 0, stream>>>(cell1, cell2);
      break;
    case 256:
      SoAFunctorNoN3Pair<floatType, 256><<<numRequiredBlocks(cell1._size), 256, 0, stream>>>(cell1, cell2);
      break;
    case 512:
      SoAFunctorNoN3Pair<floatType, 512><<<numRequiredBlocks(cell1._size), 512, 0, stream>>>(cell1, cell2);
      break;
    case 1024:
      SoAFunctorNoN3Pair<floatType, 1024><<<numRequiredBlocks(cell1._size), 1024, 0, stream>>>(cell1, cell2);
      break;
    default:
      autopas::utils::ExceptionHandler::exception(std::string("cuda Kernel size not available"));
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::SoAFunctorN3Wrapper(FunctorCudaSoA<floatType> *cell1Base, hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType> *>(cell1Base);

  switch (_num_threads) {
    case 32:
      SoAFunctorN3<floatType, 32><<<numRequiredBlocks(cell1._size), 32, 0, stream>>>(cell1);
      break;
    case 64:
      SoAFunctorN3<floatType, 64><<<numRequiredBlocks(cell1._size), 64, 0, stream>>>(cell1);
      break;
    case 128:
      SoAFunctorN3<floatType, 128><<<numRequiredBlocks(cell1._size), 128, 0, stream>>>(cell1);
      break;
    case 256:
      SoAFunctorN3<floatType, 256><<<numRequiredBlocks(cell1._size), 256, 0, stream>>>(cell1);
      break;
    case 512:
      SoAFunctorN3<floatType, 512><<<numRequiredBlocks(cell1._size), 512, 0, stream>>>(cell1);
      break;
    case 1024:
      SoAFunctorN3<floatType, 1024><<<numRequiredBlocks(cell1._size), 1024, 0, stream>>>(cell1);
      break;
    default:
      autopas::utils::ExceptionHandler::exception(std::string("cuda Kernel size not available"));
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::SoAFunctorN3PairWrapper(FunctorCudaSoA<floatType> *cell1Base,
                                                              FunctorCudaSoA<floatType> *cell2Base,
                                                              hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType> *>(cell1Base);
  LJFunctorCudaSoA<floatType> cell2 = *static_cast<LJFunctorCudaSoA<floatType> *>(cell2Base);

  switch (_num_threads) {
    case 32:
      SoAFunctorN3Pair<floatType, 32><<<numRequiredBlocks(cell1._size), 32, 0, stream>>>(cell1, cell2);
      break;
    case 64:
      SoAFunctorN3Pair<floatType, 64><<<numRequiredBlocks(cell1._size), 64, 0, stream>>>(cell1, cell2);
      break;
    case 128:
      SoAFunctorN3Pair<floatType, 128><<<numRequiredBlocks(cell1._size), 128, 0, stream>>>(cell1, cell2);
      break;
    case 256:
      SoAFunctorN3Pair<floatType, 256><<<numRequiredBlocks(cell1._size), 256, 0, stream>>>(cell1, cell2);
      break;
    case 512:
      SoAFunctorN3Pair<floatType, 512><<<numRequiredBlocks(cell1._size), 512, 0, stream>>>(cell1, cell2);
      break;
    case 1024:
      SoAFunctorN3Pair<floatType, 1024><<<numRequiredBlocks(cell1._size), 1024, 0, stream>>>(cell1, cell2);
      break;
    default:
      autopas::utils::ExceptionHandler::exception(std::string("cuda Kernel size not available"));
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <typename floatType, int block_size>
__global__ void LinkedCellsTraversalNoN3(LJFunctorCudaSoA<floatType> cell, unsigned int *cids, size_t *cellSizes,
                                         unsigned int offsets_size, int *offsets) {
  int own_cid = cids[blockIdx.x];
  __shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
  typename vec3<floatType>::Type myposition = {getInfinity<floatType>(), getInfinity<floatType>(),
                                               getInfinity<floatType>()};
  typename vec3<floatType>::Type myf = {0, 0, 0};

  int index = cellSizes[own_cid] + threadIdx.x;
  if (threadIdx.x < (cellSizes[own_cid + 1] - cellSizes[own_cid])) {
    myposition.x = cell._posX[index];
    myposition.y = cell._posY[index];
    myposition.z = cell._posZ[index];
  }
  // other cells
  for (auto other_index = 0; other_index < offsets_size; ++other_index) {
    const int other_id = own_cid + offsets[other_index];
    const int cell2Start = cellSizes[other_id];
    const int sizeCell2 = cellSizes[other_id + 1] - cell2Start;

    cell2_pos_shared[threadIdx.x] = {cell._posX[cell2Start + threadIdx.x], cell._posY[cell2Start + threadIdx.x],
                                     cell._posZ[cell2Start + threadIdx.x]};
    __syncthreads();
    for (int j = 0; j < sizeCell2; ++j) {
      myf = bodyBodyF<floatType>(myposition, cell2_pos_shared[j], myf);
    }
    __syncthreads();
  }
  if (threadIdx.x < (cellSizes[own_cid + 1] - cellSizes[own_cid])) {
    atomicAdd(cell._forceX + index, myf.x);
    atomicAdd(cell._forceY + index, myf.y);
    atomicAdd(cell._forceZ + index, myf.z);
  }
}

template <typename floatType, int block_size>
__global__ void LinkedCellsTraversalN3(LJFunctorCudaSoA<floatType> cell, unsigned int *cids, size_t *cellSizes,
                                       unsigned int offsets_size, int *offsets) {
  int own_cid = cids[blockIdx.x];
  __shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
  __shared__ typename vec3<floatType>::Type cell2_forces_shared[block_size];

  typename vec3<floatType>::Type myposition = {getInfinity<floatType>(), getInfinity<floatType>(),
                                               getInfinity<floatType>()};
  typename vec3<floatType>::Type myf = {0, 0, 0};

  int index = cellSizes[own_cid] + threadIdx.x;
  if (threadIdx.x < (cellSizes[own_cid + 1] - cellSizes[own_cid])) {
    myposition.x = cell._posX[index];
    myposition.y = cell._posY[index];
    myposition.z = cell._posZ[index];
  }
  // other cells
  for (auto other_index = 0; other_index < offsets_size; ++other_index) {
    const int other_id = own_cid + offsets[other_index];
    const int cell2Start = cellSizes[other_id];
    const int sizeCell2 = cellSizes[other_id + 1] - cell2Start;

    cell2_pos_shared[threadIdx.x] = {cell._posX[cell2Start + threadIdx.x], cell._posY[cell2Start + threadIdx.x],
                                     cell._posZ[cell2Start + threadIdx.x]};
    cell2_forces_shared[threadIdx.x] = {0, 0, 0};
    __syncthreads();
    for (int j = 0; j < sizeCell2; ++j) {
      const int offset = (j + threadIdx.x) % sizeCell2;
      myf = bodyBodyFN3<floatType, false>(myposition, cell2_pos_shared[offset], myf, cell2_forces_shared + offset);
    }
    __syncthreads();

    atomicAdd(cell._forceX + cell2Start + threadIdx.x, cell2_forces_shared[threadIdx.x].x);
    atomicAdd(cell._forceY + cell2Start + threadIdx.x, cell2_forces_shared[threadIdx.x].y);
    atomicAdd(cell._forceZ + cell2Start + threadIdx.x, cell2_forces_shared[threadIdx.x].z);
    __syncthreads();
  }
  // same cells
  {
    const int cell1Start = cellSizes[own_cid];
    const int sizeCell1 = cellSizes[own_cid + 1] - cell1Start;

    cell2_pos_shared[threadIdx.x] = {cell._posX[cell1Start + threadIdx.x], cell._posY[cell1Start + threadIdx.x],
                                     cell._posZ[cell1Start + threadIdx.x]};
    __syncthreads();
    for (int j = 0; j < sizeCell1; ++j) {
      myf = bodyBodyF<floatType>(myposition, cell2_pos_shared[j], myf);
    }
    __syncthreads();
  }
  if (threadIdx.x < (cellSizes[own_cid + 1] - cellSizes[own_cid])) {
    atomicAdd(cell._forceX + index, myf.x);
    atomicAdd(cell._forceY + index, myf.y);
    atomicAdd(cell._forceZ + index, myf.z);
  }
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::LinkedCellsTraversalNoN3Wrapper(FunctorCudaSoA<floatType> *cell1Base,
                                                                      unsigned int reqThreads, unsigned int cids_size,
                                                                      unsigned int *cids, unsigned int cellSizes_size,
                                                                      size_t *cellSizes, unsigned int offsets_size,
                                                                      int *offsets, hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType> *>(cell1Base);

  switch (reqThreads) {
    case 32:
      LinkedCellsTraversalNoN3<floatType, 32>
          <<<cids_size, 32, 0, stream>>>(cell1, cids, cellSizes, offsets_size, offsets);
      break;
    case 64:
      LinkedCellsTraversalNoN3<floatType, 64>
          <<<cids_size, 64, 0, stream>>>(cell1, cids, cellSizes, offsets_size, offsets);
      break;
    case 96:
      LinkedCellsTraversalNoN3<floatType, 96>
          <<<cids_size, 96, 0, stream>>>(cell1, cids, cellSizes, offsets_size, offsets);
      break;
    default:
      autopas::utils::ExceptionHandler::exception(
          "Linked Cells NoN3: cuda Kernel size not available for Linked cells available 32, 64, 96. Too many particles "
          "in a cell. Requested: {}",
          reqThreads);
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <typename floatType>
void LJFunctorCudaWrapper<floatType>::LinkedCellsTraversalN3Wrapper(FunctorCudaSoA<floatType> *cell1Base,
                                                                    unsigned int reqThreads, unsigned int cids_size,
                                                                    unsigned int *cids, unsigned int cellSizes_size,
                                                                    size_t *cellSizes, unsigned int offsets_size,
                                                                    int *offsets, hipStream_t stream) {
  LJFunctorCudaSoA<floatType> cell1 = *static_cast<LJFunctorCudaSoA<floatType> *>(cell1Base);

  switch (reqThreads) {
    case 32:
      LinkedCellsTraversalN3<floatType, 32>
          <<<cids_size, 32, 0, stream>>>(cell1, cids, cellSizes, offsets_size, offsets);
      break;
    case 64:
      LinkedCellsTraversalN3<floatType, 64>
          <<<cids_size, 64, 0, stream>>>(cell1, cids, cellSizes, offsets_size, offsets);
      break;
    case 96:
      LinkedCellsTraversalN3<floatType, 96>
          <<<cids_size, 96, 0, stream>>>(cell1, cids, cellSizes, offsets_size, offsets);
      break;
    default:
      autopas::utils::ExceptionHandler::exception(
          "Linked Cells N3:cuda Kernel size not available for Linked cells available 32, 64, 96. Too many particles in "
          "a cell. Requested: {}",
          reqThreads);
      break;
  }
  autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

template <>
void LJFunctorCudaWrapper<float>::loadConstants(FunctorCudaConstants<float> *constants) {
  LJFunctorConstants<float> *c = static_cast<LJFunctorConstants<float> *>(constants);

  hipMemcpyToSymbol(HIP_SYMBOL(global_constants_float), c, sizeof(LJFunctorConstants<float>));
}
template <>
void LJFunctorCudaWrapper<double>::loadConstants(FunctorCudaConstants<double> *constants) {
  LJFunctorConstants<double> *c = static_cast<LJFunctorConstants<double> *>(constants);

  hipMemcpyToSymbol(HIP_SYMBOL(global_constants_double), c, sizeof(LJFunctorConstants<double>));
}

template <typename T>
void LJFunctorCudaWrapper<T>::loadConstants(FunctorCudaConstants<T> *constants) {
  autopas::utils::ExceptionHandler::exception("Cuda constants with unknown Type loaded");
}

template class LJFunctorCudaWrapper<float>;
template class LJFunctorCudaWrapper<double>;

}  // namespace autopas
