#include "hip/hip_runtime.h"
/**
 * @file LJFunctorCuda.h
 *
 * @date 9.1.2019
 * @author jspahl
 */
#include "autopas/pairwiseFunctors/LJFunctorCuda.cuh"
#include <iostream>
#include "autopas/utils/ExceptionHandler.h"
#include "autopas/utils/CudaExceptionHandler.h"
#include "hip/hip_math_constants.h"
#include "autopas/utils/CudaDeviceVector.h"
#include "autopas/utils/CudaSoA.h"

__constant__ constants global_constants;

template<typename floating_precision = double>
class soa {
public:
	floating_precision* posX;
	floating_precision* posY;
	floating_precision* posZ;
	floating_precision* forceX;
	floating_precision* forceY;
	floating_precision* forceZ;
};

template<typename T> struct vec3 {
	typedef T Type;
};
template<> struct vec3<float> {
	typedef float3 Type;
};
template<> struct vec3<double> {
	typedef double3 Type;
};

template<typename floatType>
__device__
typename vec3<floatType>::Type bodyBodyF(typename vec3<floatType>::Type i,
		typename vec3<floatType>::Type j, typename vec3<floatType>::Type fi) {
	auto drx = i.x - j.x;
	auto dry = i.y - j.y;
	auto drz = i.z - j.z;

	auto dr2 = drx * drx + dry * dry + drz * drz;

	if (dr2 > global_constants.cutoffsquare | dr2 == 0.0) {
		return fi;
	}

	auto invdr2 = 1. / dr2;
	auto lj6 = global_constants.sigmasquare * invdr2;
	lj6 = lj6 * lj6 * lj6;
	auto lj12 = lj6 * lj6;
	auto lj12m6 = lj12 - lj6;
	auto fac = global_constants.epsilon24 * (lj12 + lj12m6) * invdr2;

	fi.x += drx * fac;
	fi.y += dry * fac;
	fi.z += drz * fac;

	return fi;
}

template<typename floatType>
__device__
typename vec3<floatType>::Type bodyBodyFN3(typename vec3<floatType>::Type i,
		typename vec3<floatType>::Type j, typename vec3<floatType>::Type fi,
		typename vec3<floatType>::Type* fj) {
	auto drx = i.x - j.x;
	auto dry = i.y - j.y;
	auto drz = i.z - j.z;

	auto dr2 = drx * drx + dry * dry + drz * drz;

	if (dr2 > global_constants.cutoffsquare) {
		return fi;
	}

	auto invdr2 = 1. / dr2;
	auto lj6 = global_constants.sigmasquare * invdr2;
	lj6 = lj6 * lj6 * lj6;
	auto lj12 = lj6 * lj6;
	auto lj12m6 = lj12 - lj6;
	auto fac = global_constants.epsilon24 * (lj12 + lj12m6) * invdr2;

	auto dfx = drx * fac;
	auto dfy = dry * fac;
	auto dfz = drz * fac;

	fi.x += dfx;
	fi.y += dfy;
	fi.z += dfz;

	atomicAdd(&(fj->x), -dfx);
	atomicAdd(&(fj->y), -dfy);
	atomicAdd(&(fj->z), -dfz);

	return fi;
}

__device__ double3 bodyBodyFcalcGlobals(double3 i, double3 j, double3 fi,
		double4 globals) {
	double drx = i.x - j.x;
	double dry = i.y - j.y;
	double drz = i.z - j.z;

	double dr2 = drx * drx + dry * dry + drz * drz;

	if (dr2 > global_constants.cutoffsquare | dr2 == 0.0) {
		return fi;
	}

	double invdr2 = 1. / dr2;
	double lj6 = global_constants.sigmasquare * invdr2;
	lj6 = lj6 * lj6 * lj6;
	double lj12 = lj6 * lj6;
	double lj12m6 = lj12 - lj6;
	double fac = global_constants.epsilon24 * (lj12 + lj12m6) * invdr2;

	const double fx = drx * fac;
	const double fy = dry * fac;
	const double fz = drz * fac;

	const double virialx = drx * fx;
	const double virialy = dry * fy;
	const double virialz = drz * fz;
	const double upot = (global_constants.epsilon24 * lj12m6
			+ global_constants.shift6);

	// these calculations assume that this functor is not called for halo cells!
	globals.w += upot;
	globals.x += virialx;
	globals.y += virialy;
	globals.z += virialz;

	fi.x += fx;
	fi.y += fy;
	fi.z += fz;

	return fi;
}

template<typename floatType, int block_size>
__global__
void SoAFunctorNoN3(int N, floatType* posX, floatType* posY, floatType* posZ,
		floatType* forceX, floatType* forceY, floatType* forceZ) {
	__shared__ typename vec3<floatType>::Type block_pos[block_size];
	int i, tile;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	typename vec3<floatType>::Type myposition = { 0, 0, 0 };
	typename vec3<floatType>::Type myf = { 0, 0, 0 };
	if (tid < N) {
		myposition.x = posX[tid];
		myposition.y = posY[tid];
		myposition.z = posZ[tid];
	}

	for (i = block_size, tile = 0; i < N; i += block_size, ++tile) {
		int idx = tile * block_size + threadIdx.x;

		block_pos[threadIdx.x] = {posX[idx], posY[idx], posZ[idx]};
		__syncthreads();
		if (tid < N) {
			for (int j = 0; j < blockDim.x; ++j) {
				myf = bodyBodyF<double>(myposition, block_pos[j], myf);
			}
		}
		__syncthreads();
	}
	{
		int idx = tile * block_size + threadIdx.x;
		block_pos[threadIdx.x] = {posX[idx], posY[idx], posZ[idx]};
		__syncthreads();

		const int size = N - tile * blockDim.x;
		for (int j = 0; j < size; ++j) {
			myf = bodyBodyF<floatType>(myposition, block_pos[j], myf);
		}

		__syncthreads();
	}

	atomicAdd(forceX + tid, myf.x);
	atomicAdd(forceY + tid, myf.y);
	atomicAdd(forceZ + tid, myf.z);
}

template<typename floatType, int block_size>
__global__
void SoAFunctorNoN3Pair(int N, floatType* posX, floatType* posY,
		floatType* posZ, floatType* forceX, floatType* forceY,
		floatType* forceZ, int M, floatType* posX2, floatType* posY2,
		floatType* posZ2) {
	__shared__ typename vec3<floatType>::Type block_pos[block_size];
	int i, tile;
	int tid = blockIdx.x * block_size + threadIdx.x;
	typename vec3<floatType>::Type myposition;
	typename vec3<floatType>::Type myf = { 0, 0, 0 };

	if (tid < N) {
		myposition.x = posX[tid];
		myposition.y = posY[tid];
		myposition.z = posZ[tid];
	}

	for (i = 0, tile = 0; i < M; i += block_size, ++tile) {
		int idx = tile * block_size + threadIdx.x;

		if (idx < M)
			block_pos[threadIdx.x] = {posX2[idx], posY2[idx], posZ2[idx]};
		__syncthreads();

		const int size = min(block_size, M - i);
		for (int j = 0; j < size; ++j) {
			myf = bodyBodyF<floatType>(myposition, block_pos[j], myf);
		}
		__syncthreads();
	}
	atomicAdd(forceX + tid, myf.x);
	atomicAdd(forceY + tid, myf.y);
	atomicAdd(forceZ + tid, myf.z);
}

template<typename floatType, int block_size, bool NMisMultipleBlockSize = false>
__global__
void SoAFunctorN3(int N, double* posX, double* posY, double* posZ,
		double* forceX, double* forceY, double* forceZ) {
	static_assert((block_size & (block_size - 1)) == 0, "block size must be power of 2");
	__shared__ typename vec3<floatType>::Type cell1_pos_shared[block_size];
	__shared__ typename vec3<floatType>::Type cell1_forces_shared[block_size];
	int tid = blockIdx.x * block_size + threadIdx.x;
	typename vec3<floatType>::Type myposition = { HIP_INF, HIP_INF,
			HIP_INF };
	typename vec3<floatType>::Type myf = { 0, 0, 0 };
	int i, tile;
	const int mask = block_size - 1;

	if (not NMisMultipleBlockSize && tid < N) {
		myposition.x = posX[tid];
		myposition.y = posY[tid];
		myposition.z = posZ[tid];
	}

	for (i = 0, tile = 0; tile < blockIdx.x; i += block_size, ++tile) {
		int idx = tile * block_size + threadIdx.x;
		cell1_pos_shared[threadIdx.x] = {posX[idx], posY[idx], posZ[idx]};
		cell1_forces_shared[threadIdx.x] = {0,0,0};
		__syncthreads();

		for (int j = 0; j < block_size; ++j) {
			const int offset = (j + threadIdx.x) & mask;
			myf = bodyBodyFN3<floatType>(myposition, cell1_pos_shared[offset],
					myf, cell1_forces_shared + offset);
		}
		__syncthreads();

		atomicAdd(forceX + idx, cell1_forces_shared[threadIdx.x].x);
		atomicAdd(forceY + idx, cell1_forces_shared[threadIdx.x].y);
		atomicAdd(forceZ + idx, cell1_forces_shared[threadIdx.x].z);
		__syncthreads();
	}

	{
		int idx = blockIdx.x * block_size + threadIdx.x;
		cell1_pos_shared[threadIdx.x] = {posX[idx], posY[idx], posZ[idx]};
		cell1_forces_shared[threadIdx.x] = {0,0,0};
		__syncthreads();

		for (int j = threadIdx.x -1; j >= 0; --j) {
			myf = bodyBodyFN3<floatType>(myposition, cell1_pos_shared[j], myf,
					cell1_forces_shared + j);
		}
		__syncthreads();

		atomicAdd(forceX + idx, cell1_forces_shared[threadIdx.x].x);
		atomicAdd(forceY + idx, cell1_forces_shared[threadIdx.x].y);
		atomicAdd(forceZ + idx, cell1_forces_shared[threadIdx.x].z);
		__syncthreads();
	}

	atomicAdd(forceX + tid, myf.x);
	atomicAdd(forceY + tid, myf.y);
	atomicAdd(forceZ + tid, myf.z);
}

template<typename floatType, int block_size, bool NMisMultipleBlockSize = false>
__global__
void SoAFunctorN3Pair(int N, soa<> cell1, int M, soa<> cell2) {
	static_assert((block_size & (block_size - 1)) == 0, "block size must be power of 2");
	__shared__ typename vec3<floatType>::Type cell2_pos_shared[block_size];
	__shared__ typename vec3<floatType>::Type cell2_forces_shared[block_size];
	int tid = blockIdx.x * block_size + threadIdx.x;
	typename vec3<floatType>::Type myposition = { HIP_INF, HIP_INF,
			HIP_INF };
	typename vec3<floatType>::Type myf = { 0, 0, 0 };
	int i, tile;
	const int mask = block_size - 1;

	if (not NMisMultipleBlockSize && tid < N) {
		myposition.x = cell1.posX[tid];
		myposition.y = cell1.posY[tid];
		myposition.z = cell1.posZ[tid];
	}

	for (i = block_size, tile = 0; i <= M; i += block_size, ++tile) {
		int idx = tile * block_size + threadIdx.x;
		cell2_pos_shared[threadIdx.x] = {cell2.posX[idx], cell2.posY[idx], cell2.posZ[idx]};
		cell2_forces_shared[threadIdx.x] = {0,0,0};
		__syncthreads();

		for (int j = 0; j < block_size; ++j) {
			const int offset = (j + threadIdx.x) & mask;
			myf = bodyBodyFN3<floatType>(myposition, cell2_pos_shared[offset],
					myf, cell2_forces_shared + offset);
		}
		__syncthreads();

		atomicAdd(cell2.forceX + idx, cell2_forces_shared[threadIdx.x].x);
		atomicAdd(cell2.forceY + idx, cell2_forces_shared[threadIdx.x].y);
		atomicAdd(cell2.forceZ + idx, cell2_forces_shared[threadIdx.x].z);
		__syncthreads();
	}
	if (not NMisMultipleBlockSize && i > M) {
		int idx = tile * block_size + threadIdx.x;
		if (idx < M) {
			cell2_pos_shared[threadIdx.x] = {cell2.posX[idx], cell2.posY[idx], cell2.posZ[idx]};
			cell2_forces_shared[threadIdx.x] = {0,0,0};
		}
		__syncthreads();

		const int size = block_size + M - i;
		for (int j = 0; j < size; ++j) {
			const int offset = (j + threadIdx.x) % size;
			myf = bodyBodyFN3<floatType>(myposition, cell2_pos_shared[offset],
					myf, cell2_forces_shared + offset);
		}
		__syncthreads();
		if (idx < M) {
			atomicAdd(cell2.forceX + idx, cell2_forces_shared[threadIdx.x].x);
			atomicAdd(cell2.forceY + idx, cell2_forces_shared[threadIdx.x].y);
			atomicAdd(cell2.forceZ + idx, cell2_forces_shared[threadIdx.x].z);
			__syncthreads();
		}
	}

	atomicAdd(cell1.forceX + tid, myf.x);
	atomicAdd(cell1.forceY + tid, myf.y);
	atomicAdd(cell1.forceZ + tid, myf.z);
}

void CudaWrapper::SoAFunctorNoN3Wrapper(int N, double* posX, double* posY,
		double* posZ, double* forceX, double* forceY, double* forceZ,
		hipStream_t stream) {
	switch (_num_threads) {
	case 32:
		SoAFunctorNoN3<double, 32> <<<numRequiredBlocks(N), 32>>>(N, posX, posY,
				posZ, forceX, forceY, forceZ);
		break;
	case 64:
		SoAFunctorNoN3<double, 64> <<<numRequiredBlocks(N), 64>>>(N, posX, posY,
				posZ, forceX, forceY, forceZ);
		break;
	case 96:
		SoAFunctorNoN3<double, 96> <<<numRequiredBlocks(N), 96>>>(N, posX, posY,
				posZ, forceX, forceY, forceZ);
		break;
	case 128:
		SoAFunctorNoN3<double, 128> <<<numRequiredBlocks(N), 128>>>(N, posX,
				posY, posZ, forceX, forceY, forceZ);
		break;
	case 256:
		SoAFunctorNoN3<double, 256> <<<numRequiredBlocks(N), 256>>>(N, posX,
				posY, posZ, forceX, forceY, forceZ);
		break;
	case 512:
		SoAFunctorNoN3<double, 512> <<<numRequiredBlocks(N), 512>>>(N, posX,
				posY, posZ, forceX, forceY, forceZ);
		break;
	case 1024:
		SoAFunctorNoN3<double, 1024> <<<numRequiredBlocks(N), 1024>>>(N, posX,
				posY, posZ, forceX, forceY, forceZ);
		break;
	default:
		autopas::utils::ExceptionHandler::exception(
				std::string("cuda Kernel size not available"));
		break;
	}
	autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

void CudaWrapper::SoAFunctorNoN3PairWrapper(int N, double* posX, double* posY,
		double* posZ, double* forceX, double* forceY, double* forceZ, int M,
		double* posX2, double* posY2, double* posZ2, hipStream_t stream) {
	switch (_num_threads) {
	case 32:
		SoAFunctorNoN3Pair<double, 32> <<<numRequiredBlocks(N), 32>>>(N, posX,
				posY, posZ, forceX, forceY, forceZ, M, posX2, posY2, posZ2);
		break;
	case 64:
		SoAFunctorNoN3Pair<double, 64> <<<numRequiredBlocks(N), 64>>>(N, posX,
				posY, posZ, forceX, forceY, forceZ, M, posX2, posY2, posZ2);
		break;
	case 96:
		SoAFunctorNoN3Pair<double, 96> <<<numRequiredBlocks(N), 96>>>(N, posX,
				posY, posZ, forceX, forceY, forceZ, M, posX2, posY2, posZ2);
		break;
	case 128:
		SoAFunctorNoN3Pair<double, 128> <<<numRequiredBlocks(N), 128>>>(N, posX,
				posY, posZ, forceX, forceY, forceZ, M, posX2, posY2, posZ2);
		break;
	case 256:
		SoAFunctorNoN3Pair<double, 256> <<<numRequiredBlocks(N), 256>>>(N, posX,
				posY, posZ, forceX, forceY, forceZ, M, posX2, posY2, posZ2);
		break;
	case 512:
		SoAFunctorNoN3Pair<double, 512> <<<numRequiredBlocks(N), 512>>>(N, posX,
				posY, posZ, forceX, forceY, forceZ, M, posX2, posY2, posZ2);
		break;
	case 1024:
		SoAFunctorNoN3Pair<double, 1024> <<<numRequiredBlocks(N), 1024>>>(N,
				posX, posY, posZ, forceX, forceY, forceZ, M, posX2, posY2,
				posZ2);
		break;
	default:
		autopas::utils::ExceptionHandler::exception(
				std::string("cuda Kernel size not available"));
		break;
	}
	autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

void CudaWrapper::SoAFunctorN3Wrapper(int N, double* posX, double* posY,
		double* posZ, double* forceX, double* forceY, double* forceZ) {

	switch (_num_threads) {
	case 32:
		SoAFunctorN3<double, 32> <<<numRequiredBlocks(N), 32>>>(N, posX, posY,
				posZ, forceX, forceY, forceZ);
		break;
	case 64:
		SoAFunctorN3<double, 64> <<<numRequiredBlocks(N), 64>>>(N, posX, posY,
				posZ, forceX, forceY, forceZ);
		break;
	case 128:
		SoAFunctorN3<double, 128> <<<numRequiredBlocks(N), 128>>>(N, posX, posY,
				posZ, forceX, forceY, forceZ);
		break;
	case 256:
		SoAFunctorN3<double, 256> <<<numRequiredBlocks(N), 256>>>(N, posX, posY,
				posZ, forceX, forceY, forceZ);
		break;
	case 512:
		SoAFunctorN3<double, 512> <<<numRequiredBlocks(N), 512>>>(N, posX, posY,
				posZ, forceX, forceY, forceZ);
		break;
	case 1024:
		SoAFunctorN3<double, 1024> <<<numRequiredBlocks(N), 1024>>>(N, posX, posY,
				posZ, forceX, forceY, forceZ);
		break;
	default:
		autopas::utils::ExceptionHandler::exception(
				std::string("cuda Kernel size not available"));
		break;
	}
	autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

void CudaWrapper::SoAFunctorN3PairWrapper(int N, double* posX, double* posY,
		double* posZ, double* forceX, double* forceY, double* forceZ, int M,
		double* posX2, double* posY2, double* posZ2, double* forceX2,
		double* forceY2, double* forceZ2) {
	soa<double> cell1;
	cell1.posX = posX;
	cell1.posY = posY;
	cell1.posZ = posZ;
	cell1.forceX = forceX;
	cell1.forceY = forceY;
	cell1.forceZ = forceZ;
	soa<double> cell2;
	cell2.posX = posX2;
	cell2.posY = posY2;
	cell2.posZ = posZ2;
	cell2.forceX = forceX2;
	cell2.forceY = forceY2;
	cell2.forceZ = forceZ2;
	switch (_num_threads) {
	case 32:
		SoAFunctorN3Pair<double, 32> <<<numRequiredBlocks(N), 32>>>(N, cell1, M,
				cell2);
		break;
	case 64:
		SoAFunctorN3Pair<double, 64> <<<numRequiredBlocks(N), 64>>>(N, cell1, M,
				cell2);
		break;
	case 128:
		SoAFunctorN3Pair<double, 128> <<<numRequiredBlocks(N), 128>>>(N, cell1,
				M, cell2);
		break;
	case 256:
		SoAFunctorN3Pair<double, 256> <<<numRequiredBlocks(N), 256>>>(N, cell1,
				M, cell2);
		break;
	case 512:
		SoAFunctorN3Pair<double, 512> <<<numRequiredBlocks(N), 512>>>(N, cell1,
				M, cell2);
		break;
	case 1024:
		SoAFunctorN3Pair<double, 1024> <<<numRequiredBlocks(N), 1024>>>(N,
				cell1, M, cell2);
		break;
	default:
		autopas::utils::ExceptionHandler::exception(
				std::string("cuda Kernel size not available"));
		break;
	}
	autopas::utils::CudaExceptionHandler::checkLastCudaCall();
}

void CudaWrapper::loadConstants(double cutoffsquare, double epsilon24,
		double sigmasquare) {

	constants c;
	c.cutoffsquare = cutoffsquare;
	c.epsilon24 = epsilon24;
	c.sigmasquare = sigmasquare;

	hipMemcpyToSymbol(HIP_SYMBOL(global_constants), &c, sizeof(constants));
}
