
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv){
    hipDeviceProp_t dP;

    int rc = hipGetDeviceProperties(&dP, 0);
    if(rc != hipSuccess) {
        hipError_t error = hipGetLastError();
        printf("CUDA error: %s", hipGetErrorString(error));
        return rc; /* Failure */
    }

    printf("%d%d", dP.major, dP.minor);
    return 0;
}
 
